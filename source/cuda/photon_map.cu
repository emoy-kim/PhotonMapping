#include "hip/hip_runtime.h"
#include "cuda/photon_map.cuh"

#ifdef USE_CUDA
namespace cuda
{
   __host__ __device__ __forceinline__
   Mat inverse(const Mat& m)
   {
      const float coef00 = m.c2.z * m.c3.w - m.c3.z * m.c2.w;
      const float coef02 = m.c1.z * m.c3.w - m.c3.z * m.c1.w;
      const float coef03 = m.c1.z * m.c2.w - m.c2.z * m.c1.w;

      const float coef04 = m.c2.y * m.c3.w - m.c3.y * m.c2.w;
      const float coef06 = m.c1.y * m.c3.w - m.c3.y * m.c1.w;
      const float coef07 = m.c1.y * m.c2.w - m.c2.y * m.c1.w;

      const float coef08 = m.c2.y * m.c3.z - m.c3.y * m.c2.z;
      const float coef10 = m.c1.y * m.c3.z - m.c3.y * m.c1.z;
      const float coef11 = m.c1.y * m.c2.z - m.c2.y * m.c1.z;

      const float coef12 = m.c2.x * m.c3.w - m.c3.x * m.c2.w;
      const float coef14 = m.c1.x * m.c3.w - m.c3.x * m.c1.w;
      const float coef15 = m.c1.x * m.c2.w - m.c2.x * m.c1.w;

      const float coef16 = m.c2.x * m.c3.z - m.c3.x * m.c2.z;
      const float coef18 = m.c1.x * m.c3.z - m.c3.x * m.c1.z;
      const float coef19 = m.c1.x * m.c2.z - m.c2.x * m.c1.z;

      const float coef20 = m.c2.x * m.c3.y - m.c3.x * m.c2.y;
      const float coef22 = m.c1.x * m.c3.y - m.c3.x * m.c1.y;
      const float coef23 = m.c1.x * m.c2.y - m.c2.x * m.c1.y;

      const float4 fac0 = make_float4( coef00, coef00, coef02, coef03 );
      const float4 fac1 = make_float4( coef04, coef04, coef06, coef07 );
      const float4 fac2 = make_float4( coef08, coef08, coef10, coef11 );
      const float4 fac3 = make_float4( coef12, coef12, coef14, coef15 );
      const float4 fac4 = make_float4( coef16, coef16, coef18, coef19 );
      const float4 fac5 = make_float4( coef20, coef20, coef22, coef23 );

      const float4 vec0 = make_float4( m.c1.x, m.c0.x, m.c0.x, m.c0.x );
      const float4 vec1 = make_float4( m.c1.y, m.c0.y, m.c0.y, m.c0.y );
      const float4 vec2 = make_float4( m.c1.z, m.c0.z, m.c0.z, m.c0.z );
      const float4 vec3 = make_float4( m.c1.w, m.c0.w, m.c0.w, m.c0.w );

      const float4 inv0 = vec1 * fac0 - vec2 * fac1 + vec3 * fac2;
      const float4 inv1 = vec0 * fac0 - vec2 * fac3 + vec3 * fac4;
      const float4 inv2 = vec0 * fac1 - vec1 * fac3 + vec3 * fac5;
      const float4 inv3 = vec0 * fac2 - vec1 * fac4 + vec2 * fac5;

      const float4 sign_a = make_float4( 1.0f, -1.0f, 1.0f, -1.0f );
      const float4 sign_b = make_float4( -1.0f, 1.0f, -1.0f, 1.0f );
      Mat inv(inv0 * sign_a, inv1 * sign_b, inv2 * sign_a, inv3 * sign_b);
      const float one_over_det = 1.0f / dot( m.c0, make_float4( inv.c0.x, inv.c1.x, inv.c2.x, inv.c3.x ) );
      inv.c0 *= one_over_det;
      inv.c1 *= one_over_det;
      inv.c2 *= one_over_det;
      inv.c3 *= one_over_det;
      return inv;
   }

   __host__ __device__ __forceinline__
   Mat transpose(const Mat& m)
   {
      Mat t;
      t.c0.x = m.c0.x;
      t.c0.y = m.c1.x;
      t.c0.z = m.c2.x;
      t.c0.w = m.c3.x;

      t.c1.x = m.c0.y;
      t.c1.y = m.c1.y;
      t.c1.z = m.c2.y;
      t.c1.w = m.c3.y;

      t.c2.x = m.c0.z;
      t.c2.y = m.c1.z;
      t.c2.z = m.c2.z;
      t.c2.w = m.c3.z;

      t.c3.x = m.c0.w;
      t.c3.y = m.c1.w;
      t.c3.z = m.c2.w;
      t.c3.w = m.c3.w;
      return t;
   }

   __host__ __device__ __forceinline__
   float3 transform(const Mat& m, const float3& v)
   {
      return make_float3(
         m.c0.x * v.x + m.c1.x * v.y + m.c2.x * v.z + m.c3.x,
         m.c0.y * v.x + m.c1.y * v.y + m.c2.y * v.z + m.c3.y,
         m.c0.z * v.x + m.c1.z * v.y + m.c2.z * v.z + m.c3.z
      );
   }

   __host__ __device__ __forceinline__
   Mat getVectorTransform(const Mat& m)
   {
      return transpose( inverse( m ) );
   }

   __host__ __device__ __forceinline__
   float3 transformVector(const Mat& m, const float3& v)
   {
      return make_float3(
         m.c0.x * v.x + m.c1.x * v.y + m.c2.x * v.z,
         m.c0.y * v.x + m.c1.y * v.y + m.c2.y * v.z,
         m.c0.z * v.x + m.c1.z * v.y + m.c2.z * v.z
      );
   }

   __device__ __forceinline__
   float getRandomValue(hiprandState* state, float a, float b)
   {
      float r = hiprand_uniform( state );
      r = r * (b - a) + a;
      return r;
   }

   // this hemisphere is towards the y-axis, and its lower plane is the xz-plane.
   __device__
   float3 getRandomPointInUnitHemisphere(float& pdf, hiprandState* state)
   {
      const float phi = 2.0f * HIP_PI_F * getRandomValue( state, 0.0f, 1.0f ); // [0, 2pi]
      const float theta = acos( getRandomValue( state, -1.0f, 1.0f ) ) * 0.5f; // [0, pi/2]
      const float cos_theta = cos( theta );
      pdf *= cos_theta * HIP_2_OVER_PI_F * 0.5f;
      return { cos( phi ) * sin( theta ), cos_theta, sin( phi ) * sin( theta ) };
   }

   __device__
   float3 getSamplePointAroundAxis(float& pdf, const float3& v, hiprandState* state)
   {
      const float3 u = abs( v.y ) < 0.9f ?
         normalize( cross( v, make_float3( 0.0f, 1.0f, 0.0f ) ) ) :
         normalize( cross( v, make_float3( 0.0f, 0.0f, 1.0f ) ) );
      const float3 n = normalize( cross( u, v ) );
      const float3 p = getRandomPointInUnitHemisphere( pdf, state );
      return make_float3(
         u.x * p.x + v.x * p.y + n.x * p.z,
         u.y * p.x + v.y * p.y + n.y * p.z,
         u.z * p.x + v.z * p.y + n.z * p.z
      );
   }

   __device__
   float3 getSampleRayFromLight(
      float3& ray_origin,
      float3& ray_direction,
      hiprandState* state,
      const AreaLight* lights
   )
   {
      // Currently, the number of lights is 2.
      const int light_index = getRandomValue( state, 0.0f, 1.0f ) > 0.5f ? 0 : 1;
      const float3 v0 = lights[light_index].Vertex0;
      const float3 v1 = lights[light_index].Vertex1;
      const float3 v2 = lights[light_index].Vertex2;
      const float a = getRandomValue( state, 0.0f, 1.0f );
      const float b = getRandomValue( state, 0.0f, 1.0f );
      ray_origin = (1.0f - a - b) * v0 + a * v1 + b * v2;
      ray_origin = transform( lights[light_index].ToWorld, ray_origin );

      const float3 normal = lights[light_index].Normal;
      float pdf = 1.0f / (lights[light_index].Area * 2.0f);
      ray_direction = getSamplePointAroundAxis( pdf, normal, state );
      const float3 power = lights[light_index].Emission / pdf * abs( dot( ray_direction, normal ) );

      const Mat n = getVectorTransform( lights[light_index].ToWorld );
      ray_direction = transformVector( n, ray_direction );
      return power;
   }

   __device__
   bool intersectWithBox(const float3& ray_origin, const float3& ray_direction, const Box& box)
   {
      float exit = HIP_INF_F;
      float enter = -HIP_INF_F;
      if (abs( ray_direction.x ) > 1e-5f) {
         const float t_min = (box.MinPoint.x - ray_origin.x) / ray_direction.x;
         const float t_max = (box.MaxPoint.x - ray_origin.x) / ray_direction.x;
         const float t_enter = min( t_min, t_max );
         const float t_exit = max( t_min, t_max );
         if (t_enter > enter) enter = t_enter;
         if (t_exit < exit) exit = t_exit;
         if (enter > exit || exit < 0.0f) return false;
      }
      else if (ray_origin.x < box.MinPoint.x || box.MaxPoint.x < ray_origin.x) return false;

      if (abs( ray_direction.y ) > 1e-5f) {
         const float t_min = (box.MinPoint.y - ray_origin.y) / ray_direction.y;
         const float t_max = (box.MaxPoint.y - ray_origin.y) / ray_direction.y;
         const float t_enter = min( t_min, t_max );
         const float t_exit = max( t_min, t_max );
         if (t_enter > enter) enter = t_enter;
         if (t_exit < exit) exit = t_exit;
         if (enter > exit || exit < 0.0f) return false;
      }
      else if (ray_origin.y < box.MinPoint.y || box.MaxPoint.y < ray_origin.y) return false;

      if (abs( ray_direction.z ) > 1e-5f) {
         const float t_min = (box.MinPoint.z - ray_origin.z) / ray_direction.z;
         const float t_max = (box.MaxPoint.z - ray_origin.z) / ray_direction.z;
         const float t_enter = min( t_min, t_max );
         const float t_exit = max( t_min, t_max );
         if (t_enter > enter) enter = t_enter;
         if (t_exit < exit) exit = t_exit;
         if (enter > exit || exit < 0.0f) return false;
      }
      else if (ray_origin.z < box.MinPoint.z || box.MaxPoint.z < ray_origin.z) return false;

      //distance = enter > 0.0f ? enter : exit;
      return true;
   }

   __device__
   bool intersectWithTriangle(
      float3& tuv,
      const float3& ray_origin,
      const float3& ray_direction,
      const float3& p0,
      const float3& p1,
      const float3& p2
   )
   {
      const float3 e1 = p1 - p0;
      const float3 e2 = p2 - p0;
      const float3 q = cross( ray_direction, e2 );
      const float det = dot( e1, q );
      if (abs( det ) < 1e-5f) return false;

      const float f = 1.0f / det;
      const float3 s = ray_origin - p0;
      const float u = f * dot( s, q );
      if (u < 0.0f) return false;

      const float3 r = cross( s, e1 );
      const float v = f * dot( ray_direction, r );
      if (v < 0.0f || u + v > 1.0f) return false;

      const float t = f * dot( e2, r );
      if (t <= 0.0f) return false;

      tuv = make_float3( t, u, v );
      return true;
   }

   __device__
   bool findIntersection(
      IntersectionInfo& intersection,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      const float3& ray_origin,
      const float3& ray_direction,
      int object_num
   )
   {
      bool intersect = false;
      float distance = HIP_INF_F;
      int offset = 0, index_offset = 0;
      for (int i = 0; i < object_num; ++i) {
         if (intersectWithBox( ray_origin, ray_direction, world_bounds[i] )) {
            const Mat vector_transform = getVectorTransform( to_worlds[i] );
            for (int j = 0; j < index_sizes[i]; j += 3) {
               const int k0 = offset + indices[index_offset + j];
               const int k1 = offset + indices[index_offset + j + 1];
               const int k2 = offset + indices[index_offset + j + 2];
               const float3 p0 = transform( to_worlds[i], vertices[k0] );
               const float3 p1 = transform( to_worlds[i], vertices[k1] );
               const float3 p2 = transform( to_worlds[i], vertices[k2] );

               float3 tuv;
               if (intersectWithTriangle( tuv, ray_origin, ray_direction, p0, p1, p2 )) {
                  if (distance > tuv.x) {
                     const float3 n0 = normalize( transformVector( vector_transform, normals[k0] ) );
                     const float3 n1 = normalize( transformVector( vector_transform, normals[k1] ) );
                     const float3 n2 = normalize( transformVector( vector_transform, normals[k2] ) );
                     distance = tuv.x;
                     intersection.ObjectIndex = i;
                     intersection.Normal = (n0 + n1 + n2) / 3.0f;
                     intersection.Position = ray_origin + tuv.x * ray_direction;
                     intersection.ShadingNormal = (1.0f - tuv.y - tuv.z) * n0 + tuv.y * n1 + tuv.z * n2;
                     intersect = true;
                  }
               }
            }
         }
         offset += vertex_sizes[i];
         index_offset += index_sizes[i];
      }
      return intersect;
   }

   __device__
   float getSchlickApproximation(float cos_theta, float n1, float n2)
   {
      const float r0 = (n1 - n2) * (n1 - n2) / ((n1 + n2) * (n1 + n2));
      const float x = 1.0f - cos_theta;
      const float x2 = x * x;
      return r0 + (1.0f - r0) * x2 * x2 * x;
   }

   __device__
   float correctShadingNormal(
      const float3& wo,
      const float3& wi,
      const float3& normal,
      const float3& shading_normal
   )
   {
      const float wo_dot_n = dot( wo, normal );
      const float wi_dot_n = dot( wi, normal );
      const float wo_dot_sn = dot( wo, shading_normal );
      const float wi_dot_sn = dot( wi, shading_normal );
      if (wo_dot_n * wo_dot_sn <= 0.0f || wi_dot_n * wi_dot_sn <= 0.0f) return 0.0f;

      const float a = abs( wo_dot_n * wi_dot_sn );
      return a < 1e-5f ? 1.0f : abs( wo_dot_sn * wi_dot_n ) / a;
   }

   __device__
   float3 getSampleBRDF(
      float& pdf,
      float3& outgoing,
      const float3& ray_direction,
      hiprandState* state,
      const Material* materials,
      const IntersectionInfo& intersection,
      bool from_camera
   )
   {
      pdf = 1.0f;
      float3 brdf;
      const float3 incoming = -ray_direction;
      if (materials[intersection.ObjectIndex].MaterialType == MATERIAL_TYPE::LAMBERT) {
         outgoing = getSamplePointAroundAxis( pdf, intersection.ShadingNormal, state );
         if (dot( incoming, intersection.ShadingNormal ) < 0.0f ||
             dot( outgoing, intersection.ShadingNormal ) < 0.0f) {
            brdf = make_float3( 0.0f, 0.0f, 0.0f );
         }
         else brdf = materials[intersection.ObjectIndex].Diffuse * HIP_2_OVER_PI_F * 0.5f;
      }
      else if (materials[intersection.ObjectIndex].MaterialType == MATERIAL_TYPE::MIRROR) {
         outgoing = reflect( -incoming, intersection.ShadingNormal );
         const float d = abs( dot( outgoing, intersection.ShadingNormal ) );
         brdf = d < 1e-5f ? make_float3( 0.0f, 0.0f, 0.0f ) : make_float3( 1.0f, 1.0f, 1.0f ) / d;
      }
      else if (materials[intersection.ObjectIndex].MaterialType == MATERIAL_TYPE::GLASS) {
         float3 n;
         float in_refractive_index, out_refractive_index;
         if (dot( incoming, intersection.ShadingNormal ) > 0.0f) {
            out_refractive_index = 1.0f;
            in_refractive_index = materials[intersection.ObjectIndex].RefractiveIndex;
            n = intersection.ShadingNormal;
         }
         else {
            in_refractive_index = 1.0f;
            out_refractive_index = materials[intersection.ObjectIndex].RefractiveIndex;
            n = -intersection.ShadingNormal;
         }

         const float fresnel = getSchlickApproximation( dot( incoming, n ), out_refractive_index, in_refractive_index );
         if (getRandomValue( state, 0.0f, 1.0f ) < fresnel) {
            outgoing = reflect( -incoming, n );
            const float d = abs( dot( outgoing, n ) );
            brdf = d < 1e-5f ? make_float3( 0.0f, 0.0f, 0.0f ) : make_float3( 1.0f, 1.0f, 1.0f ) / d;
         }
         else {
            outgoing = refract( -incoming, n, out_refractive_index / in_refractive_index );
            if (outgoing.x == 0.0f && outgoing.y == 0.0f && outgoing.z == 0.0f) {
               outgoing = reflect( -incoming, n );
               const float d = abs( dot( outgoing, n ) );
               brdf = d < 1e-5f ? make_float3( 0.0f, 0.0f, 0.0f ) : make_float3( 1.0f, 1.0f, 1.0f ) / d;
            }
            else {
               const float d = abs( dot( outgoing, n ) );
               const float f = from_camera ?
                  (out_refractive_index * out_refractive_index) / (in_refractive_index * in_refractive_index) : 1.0f;
               brdf = d < 1e-5f ? make_float3( 0.0f, 0.0f, 0.0f ) : make_float3( 1.0f, 1.0f, 1.0f ) * f / d;
            }
         }
      }
      outgoing = normalize( outgoing );
      return brdf;
   }

   __global__
   void cuCreatePhotonMap(
      Photon* photons,
      const AreaLight* lights,
      const Material* materials,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      int object_num,
      uint seed
   )
   {
      int generated_num = 0;
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      const int photons_to_generate = divideUp( MaxGlobalPhotonNum, step );
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x) * photons_to_generate;

      hiprandState state;
      hiprand_init( seed, index, 0, &state );

      while (true) {
         float3 ray_origin, ray_direction;
         float3 power = getSampleRayFromLight( ray_origin, ray_direction, &state, lights );
         for (int i = 0; i < MaxDepth; ++i) {
            if (power.x < 0.0f || power.y < 0.0f || power.z < 0.0f) break;

            IntersectionInfo intersection;
            if (!findIntersection(
                  intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
                  ray_origin, ray_direction, object_num
               )) break;

            if (materials[intersection.ObjectIndex].MaterialType == MATERIAL_TYPE::LAMBERT) {
               photons[index + generated_num].Power = power;
               photons[index + generated_num].Position = intersection.Position;
               photons[index + generated_num].IncomingDirection = -ray_direction;
               generated_num++;
               if (generated_num == photons_to_generate || index + generated_num >= MaxGlobalPhotonNum) return;
            }

            if (i > 0) {
               const float russian_roulette = min( max( power.x, max( power.y, power.z ) ), 1.0f );
               if (getRandomValue( &state, 0.0f, 1.0f ) < russian_roulette) power /= russian_roulette;
               else break;
            }

            float pdf;
            float3 outgoing;
            power *= getSampleBRDF( pdf, outgoing, ray_direction, &state, materials, intersection, false );
            power *=
               correctShadingNormal( outgoing, -ray_direction, intersection.Normal, intersection.ShadingNormal ) / pdf;
            ray_origin = intersection.Position;
            ray_direction = outgoing;
         }
      }
   }

   __global__
   void cuPrepareKdtree(float* coordinates, const Photon* photons, int size)
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         coordinates[3 * i] = photons[i].Position.x;
         coordinates[3 * i + 1] = photons[i].Position.y;
         coordinates[3 * i + 2] = photons[i].Position.z;
      }
   }

   __device__
   void findNearestNeighbor(
      int& found_index,
      float& found_distance,
      const KdtreeNode* root,
      const Photon* photons,
      const float3& query,
      int node_index,
      int size
   )
   {
      constexpr int dim = 3;
      int depth = 0;
      int prev = -1;
      int curr = node_index;
      found_index = -1;
      found_distance = HIP_INF_F;
      while (curr >= 0) {
         const KdtreeNode* node = &root[curr];
         const int parent = node->ParentIndex;
         if (curr >= size) {
            prev = curr;
            curr = parent;
            continue;
         }

         const bool from_child = prev >= 0 && (prev == node->LeftChildIndex || prev == node->RightChildIndex);
         if (!from_child) {
            const float3 v = query - photons[node->Index].Position;
            const float squared_distance = dot( v, v );
            if (squared_distance <= found_distance) {
               found_index = curr;
               found_distance = squared_distance;
            }
         }

         float t;
         const int axis = depth % dim;
         if (axis == 0) t = query.x - photons[node->Index].Position.x;
         else if (axis == 1) t = query.y - photons[node->Index].Position.y;
         else t = query.z - photons[node->Index].Position.z;
         const bool right_priority = t > 0;
         const int far_child = right_priority ? node->LeftChildIndex : node->RightChildIndex;
         const int close_child = right_priority ? node->RightChildIndex : node->LeftChildIndex;

         int next = -1;
         if (prev >= 0 && prev == close_child) {
            if (far_child >= 0 && (t == 0 || t * t <= found_distance)) {
               next = far_child;
               depth++;
            }
            else {
               next = parent;
               depth--;
            }
         }
         else if (prev >= 0 && prev == far_child) {
            next = parent;
            depth--;
         }
         else if (prev < 0 || prev == parent) {
            if (close_child < 0 && far_child < 0) {
               next = parent;
               depth--;
            }
            else if (close_child < 0) {
               next = far_child;
               depth++;
            }
            else {
               next = close_child;
               depth++;
            }
         }

         prev = curr;
         curr = next;
      }
   }

   __device__
   float push(int found_index[], float found_distance[], int node_index, float squared_distance)
   {
      int n = node_index;
      float d = squared_distance;
      for (int i = 0; i < NeighborNum; ++i) {
         float max_distance = found_distance[i];
         float min_distance = d;
         int max_index = found_index[i];
         int min_index = n;
         if (max_distance < min_distance) {
            max_distance = d;
            min_distance = found_distance[i];
            max_index = n;
            min_index = found_index[i];
         }
         found_index[i] = min_index;
         found_distance[i] = min_distance;
         n = max_index;
         d = max_distance;
      }
      return found_distance[NeighborNum - 1];
   }

   __device__
   void findNearestNeighbors(
      int found_index[],
      float found_distance[],
      const KdtreeNode* root,
      const Photon* photons,
      const float3& query,
      int node_index,
      int size
   )
   {
      constexpr int dim = 3;
      int depth = 0;
      int prev = -1;
      int curr = node_index;
      float max_squared_distance = HIP_INF_F;
      while (curr >= 0) {
         const KdtreeNode* node = &root[curr];
         const int parent = node->ParentIndex;
         if (curr >= size) {
            prev = curr;
            curr = parent;
            continue;
         }

         const bool from_child = prev >= 0 && (prev == node->LeftChildIndex || prev == node->RightChildIndex);
         if (!from_child) {
            const float3 v = query - photons[node->Index].Position;
            const float squared_distance = dot( v, v );
            if (squared_distance <= max_squared_distance) {
               max_squared_distance = push( found_index, found_distance, curr, squared_distance );
            }
         }

         float t;
         const int axis = depth % dim;
         if (axis == 0) t = query.x - photons[node->Index].Position.x;
         else if (axis == 1) t = query.y - photons[node->Index].Position.y;
         else t = query.z - photons[node->Index].Position.z;
         const bool right_priority = t > 0;
         const int far_child = right_priority ? node->LeftChildIndex : node->RightChildIndex;
         const int close_child = right_priority ? node->RightChildIndex : node->LeftChildIndex;

         int next = -1;
         if (prev >= 0 && prev == close_child) {
            if (far_child >= 0 && (t == 0 || t * t <= max_squared_distance)) {
               next = far_child;
               depth++;
            }
            else {
               next = parent;
               depth--;
            }
         }
         else if (prev >= 0 && prev == far_child) {
            next = parent;
            depth--;
         }
         else if (prev < 0 || prev == parent) {
            if (close_child < 0 && far_child < 0) {
               next = parent;
               depth--;
            }
            else if (close_child < 0) {
               next = far_child;
               depth++;
            }
            else {
               next = close_child;
               depth++;
            }
         }

         prev = curr;
         curr = next;
      }
   }

   __global__
   void cuVisualizePhotonMap(
      uint8_t* image_buffer,
      const Photon* photons,
      const KdtreeNode* root,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      Mat inverse_view,
      int root_node,
      int width,
      int height,
      int object_num,
      int size
   )
   {
      const auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);
      if (x >= width || y >= height) return;

      const int k = (y * width + x) * 3;
      const auto w = static_cast<float>(width);
      const auto h = static_cast<float>(height);
      const float u = (2.0f * static_cast<float>(x) - w) / h;
      const float v = (2.0f * static_cast<float>(y) - h) / h;
      const float3 ray_origin = make_float3( inverse_view.c3.x, inverse_view.c3.y, inverse_view.c3.z );
      const float3 ray_direction = normalize( transform( inverse_view, make_float3( u, v, -1.0f ) ) - ray_origin );

      IntersectionInfo intersection;
      if (!findIntersection(
            intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
            ray_origin, ray_direction, object_num
         )) {
         image_buffer[k] = image_buffer[k + 1] = image_buffer[k + 2] = 0;
         return;
      }

      int index;
      float squared_distance;
      findNearestNeighbor( index, squared_distance, root, photons, intersection.Position, root_node, size );
      if (squared_distance < 0.1f) {
         float3 power = photons[index].Power;
         power.x = min( max( power.x * 255.0f, 0.0f ), 255.0f );
         power.y = min( max( power.y * 255.0f, 0.0f ), 255.0f );
         power.z = min( max( power.z * 255.0f, 0.0f ), 255.0f );
         image_buffer[k + 2] = static_cast<uint8_t>(power.x);
         image_buffer[k + 1] = static_cast<uint8_t>(power.y);
         image_buffer[k] = static_cast<uint8_t>(power.z);
      }
      else image_buffer[k] = image_buffer[k + 1] = image_buffer[k + 2] = 0;
   }

   __device__
   bool hitLight(float3& emission, const AreaLight* lights, const float3& ray_origin, const float3& ray_direction)
   {
      float3 tuv;
      float3 v0 = lights[0].Vertex0;
      float3 v1 = lights[0].Vertex1;
      float3 v2 = lights[0].Vertex2;
      float3 p0 = transform( lights[0].ToWorld, v0 );
      float3 p1 = transform( lights[0].ToWorld, v1 );
      float3 p2 = transform( lights[0].ToWorld, v2 );
      if (intersectWithTriangle( tuv, ray_origin, ray_direction, p0, p1, p2 )) {
         emission = lights[0].Emission;
         return true;
      }

      v0 = lights[1].Vertex0;
      v1 = lights[1].Vertex1;
      v2 = lights[1].Vertex2;
      p0 = transform( lights[1].ToWorld, v0 );
      p1 = transform( lights[1].ToWorld, v1 );
      p2 = transform( lights[1].ToWorld, v2 );
      if (intersectWithTriangle( tuv, ray_origin, ray_direction, p0, p1, p2 )) {
         emission = lights[1].Emission;
         return true;
      }
      return false;
   }

   __device__
   float3 computeRadianceWithPhotonMap(
      const IntersectionInfo& intersection,
      const KdtreeNode* root,
      const Photon* photons,
      const Material* materials,
      const float3& ray_direction,
      int root_node,
      int size
   )
   {
      float3 radiance = make_float3( 0.0f, 0.0f, 0.0f );
      if (materials[intersection.ObjectIndex].MaterialType != MATERIAL_TYPE::LAMBERT) return radiance;

      int indices[NeighborNum];
      float squared_distances[NeighborNum];
      for (int i = 0; i < NeighborNum; ++i) {
         indices[i] = -1;
         squared_distances[i] = HIP_INF_F;
      }
      findNearestNeighbor( indices[0], squared_distances[0], root, photons, intersection.Position, root_node, size );

      float max_distance = 0.0f;
      for (int i = 0; i < NeighborNum; ++i) {
         if (indices[i] < 0) break;

         if (dot( photons[indices[i]].IncomingDirection, intersection.ShadingNormal ) > 0.0f) {
            radiance +=
               photons[indices[i]].Power * materials[intersection.ObjectIndex].Diffuse * HIP_2_OVER_PI_F * 0.5f;
         }
         max_distance = squared_distances[i];
      }
      if (indices[0] >= 0) radiance /= static_cast<float>(size) * HIP_PI_F * max_distance;
      return radiance;
   }

   __device__
   float3 computeDirectIllumination(
      const IntersectionInfo& intersection,
      const AreaLight* lights,
      const Material* materials,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3& ray_origin,
      const float3& ray_direction,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      hiprandState* state,
      int object_num
   )
   {
      // Currently, the number of lights is 2.
      const int light_index = getRandomValue( state, 0.0f, 1.0f ) > 0.5f ? 0 : 1;
      const float3 v0 = lights[light_index].Vertex0;
      const float3 v1 = lights[light_index].Vertex1;
      const float3 v2 = lights[light_index].Vertex2;
      const float a = getRandomValue( state, 0.0f, 1.0f );
      const float b = getRandomValue( state, 0.0f, 1.0f );
      float3 light_position = (1.0f - a - b) * v0 + a * v1 + b * v2;
      light_position = transform( lights[light_index].ToWorld, light_position );

      const float3 normal = lights[light_index].Normal;
      float pdf = 1.0f / (lights[light_index].Area * 2.0f);
      const float3 incoming = normalize( light_position - intersection.Position );
      const float l = length( light_position - intersection.Position );
      pdf *= l * l / abs( dot( -incoming, normal ) );

      float3 shadow_ray_origin = intersection.Position;
      float3 shadow_ray_direction = incoming;

      float3 radiance = make_float3( 0.0f, 0.0f, 0.0f );
      IntersectionInfo shadow_intersection;
      if (!findIntersection(
            shadow_intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
            shadow_ray_origin, shadow_ray_direction, object_num
         )) {
         if (materials[intersection.ObjectIndex].MaterialType == MATERIAL_TYPE::LAMBERT) {
            if (dot( -ray_direction, intersection.ShadingNormal ) > 0.0f &&
                dot( incoming, intersection.ShadingNormal ) > 0.0f) {
               radiance = lights[light_index].Emission * abs( dot( incoming, intersection.ShadingNormal ) ) *
                  materials[intersection.ObjectIndex].Diffuse * HIP_2_OVER_PI_F * 0.5f / pdf;
            }
         }
      }
      return radiance;
   }

   __device__
   float3 computeIndirectIllumination(
      const IntersectionInfo& intersection,
      const KdtreeNode* root,
      const Photon* photons,
      const AreaLight* lights,
      const Material* materials,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3& ray_origin,
      const float3& ray_direction,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      hiprandState* state,
      int root_node,
      int object_num,
      int size
   )
   {
      int depth = 0;
      float3 curr_ray_direction = ray_direction;
      float3 power = make_float3( 1.0f, 1.0f, 1.0f );
      float3 radiance = make_float3( 0.0f, 0.0f, 0.0f );
      IntersectionInfo curr_info = intersection;
      while (depth < MaxDepth) {
         float pdf;
         float3 gathering_ray_direction;
         float3 gathering_ray_origin = curr_info.Position;
         power *= getSampleBRDF( pdf, gathering_ray_direction, curr_ray_direction, state, materials, curr_info, true );
         power *= abs( dot( gathering_ray_direction, curr_info.ShadingNormal ) );

         IntersectionInfo gathering_intersection;
         if (!findIntersection(
               gathering_intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
               gathering_ray_origin, gathering_ray_direction, object_num
            )) break;

         if (materials[gathering_intersection.ObjectIndex].MaterialType == MATERIAL_TYPE::LAMBERT) {
            radiance += power / pdf * computeRadianceWithPhotonMap(
               gathering_intersection, root, photons, materials, ray_direction, root_node, size
            );
            break;
         }
         else {
            curr_ray_direction = -gathering_ray_direction;
            curr_info = gathering_intersection;
            depth++;
         }
      }
      return radiance;
   }

   __device__
   float3 getRadiance(
      const Photon* photons,
      const AreaLight* lights,
      const Material* materials,
      const KdtreeNode* root,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3& ray_origin,
      const float3& ray_direction,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      hiprandState* state,
      int root_node,
      int object_num,
      int max_depth,
      int size
   )
   {
      int depth = 0;
      float3 curr_ray_origin = ray_origin;
      float3 curr_ray_direction = ray_direction;
      while (depth < MaxDepth) {
         IntersectionInfo intersection;
         if (!findIntersection(
               intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
               curr_ray_origin, curr_ray_direction, object_num
            )) return make_float3( 0.0f, 0.0f, 0.0f );

         float3 emission;
         if (hitLight( emission, lights, curr_ray_origin, curr_ray_direction )) return emission;

         if (materials[intersection.ObjectIndex].MaterialType == MATERIAL_TYPE::LAMBERT) {
            if (depth >= GatheringDepth) {
               return computeRadianceWithPhotonMap(
                  intersection, root, photons, materials, curr_ray_direction, root_node, size
               );
            }
            else {
               const float3 direct = computeDirectIllumination(
                  intersection, lights, materials, world_bounds, to_worlds, curr_ray_origin, curr_ray_direction,
                  vertices, normals, indices, vertex_sizes, index_sizes, state, object_num
               );
               //const float3 caustics = computeCausticsWithPhotonMap();
               const float3 indirect = computeIndirectIllumination(
                  intersection, root, photons, lights, materials, world_bounds, to_worlds,
                  curr_ray_origin, curr_ray_direction, vertices, normals, indices, vertex_sizes, index_sizes,
                  state, root_node, object_num, size
               );
               return direct + indirect;
            }
         }
         else {
            if (depth >= 3) {

            }
            depth++;
            break;
         }

      }
      return make_float3( 0.0f, 0.0f, 0.0f );
   }

   __global__
   void cuRenderScene(
      uint8_t* image_buffer,
      const Photon* photons,
      const KdtreeNode* root,
      const AreaLight* lights,
      const Material* materials,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      Mat inverse_view,
      int root_node,
      int width,
      int height,
      int object_num,
      int size,
      uint seed
   )
   {
      const auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);
      if (x >= width || y >= height) return;

      hiprandState state;
      hiprand_init( seed, x * 1973 + y * 9277, 0, &state );

      const auto w = static_cast<float>(width);
      const auto h = static_cast<float>(height);
      const float3 ray_origin = make_float3( inverse_view.c3.x, inverse_view.c3.y, inverse_view.c3.z );
      float3 color = make_float3( 0.0f, 0.0f, 0.0f );
      for (int s = 0; s < SampleNum; ++s) {
         const float u = (2.0f * (static_cast<float>(x) + getRandomValue( &state, 0.0f, 1.0f )) - w) / h;
         const float v = (2.0f * (static_cast<float>(y) + getRandomValue( &state, 0.0f, 1.0f )) - h) / h;
         const float3 ray_direction = normalize( transform( inverse_view, make_float3( u, v, -1.0f ) ) - ray_origin );
         color += getRadiance(
            photons, lights, materials, root, world_bounds, to_worlds, ray_origin, ray_direction,
            vertices, normals, indices, vertex_sizes, index_sizes, &state, root_node, object_num, 0, size
         );
      }
      color /= static_cast<float>(SampleNum);

      const int k = (y * width + x) * 3;
      image_buffer[k + 2] = static_cast<uint8_t>(min( max( color.x * 255.0f, 0.0f ), 255.0f ));
      image_buffer[k + 1] = static_cast<uint8_t>(min( max( color.y * 255.0f, 0.0f ), 255.0f ));
      image_buffer[k] = static_cast<uint8_t>(min( max( color.z * 255.0f, 0.0f ), 255.0f ));
   }

   PhotonMap::PhotonMap() : Device(), ObjectNum( 0 )
   {
      ViewMatrix = getViewMatrix(
         make_float3( 0.0f, 250.0f, 750.0f ),
         make_float3( 0.0f, 250.0f, 0.0f ),
         make_float3( 0.0f, 1.0f, 0.0f )
      );
      InverseViewMatrix = inverse( ViewMatrix );
   }

   PhotonMap::~PhotonMap()
   {
      if (Device.VertexPtr != nullptr) hipFree( Device.VertexPtr );
      if (Device.VertexPtr != nullptr) hipFree( Device.VertexPtr );
      if (Device.NormalPtr != nullptr) hipFree( Device.NormalPtr );
      if (Device.IndexPtr != nullptr) hipFree( Device.IndexPtr );
      if (Device.VertexSizesPtr != nullptr) hipFree( Device.VertexSizesPtr );
      if (Device.IndexSizesPtr != nullptr) hipFree( Device.IndexSizesPtr );
      if (Device.WorldBoundsPtr != nullptr) hipFree( Device.WorldBoundsPtr );
      if (Device.ToWorldsPtr != nullptr) hipFree( Device.ToWorldsPtr );
      if (Device.MaterialsPtr != nullptr) hipFree( Device.MaterialsPtr );
      if (Device.AreaLightsPtr != nullptr) hipFree( Device.AreaLightsPtr );
   }

   void PhotonMap::initialize()
   {
      assert( Vertices.size() == Normals.size() );

      ObjectNum = static_cast<int>(Materials.size());

      int device_num = 0;
      CHECK_CUDA( hipGetDeviceCount( &device_num ) );
      if( device_num <= 0 ) throw std::runtime_error( "cuda device not found\n" );

      Device.ID = 0;
      CHECK_CUDA( hipSetDevice( Device.ID ) );

      auto buffer_size = sizeof( float3 ) * Vertices.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.VertexPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.VertexPtr, Vertices.data(), buffer_size, hipMemcpyHostToDevice ) );

      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.NormalPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.NormalPtr, Normals.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( int ) * Indices.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.IndexPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.IndexPtr, Indices.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( int ) * VertexSizes.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.VertexSizesPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.VertexSizesPtr, VertexSizes.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( int ) * IndexSizes.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.IndexSizesPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.IndexSizesPtr, IndexSizes.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( Box ) * WorldBounds.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.WorldBoundsPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.WorldBoundsPtr, WorldBounds.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( Mat ) * ToWorlds.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.ToWorldsPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.ToWorldsPtr, ToWorlds.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( Material ) * Materials.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.MaterialsPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.MaterialsPtr, Materials.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( AreaLight ) * AreaLights.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.AreaLightsPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.AreaLightsPtr, AreaLights.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( Photon ) * MaxGlobalPhotonNum;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.GlobalPhotonsPtr), buffer_size ) );
   }

   void PhotonMap::createPhotonMap()
   {
      initialize();

      std::cout << ">> Create Photon Map ...\n";
      constexpr int block_num = 128;
      constexpr int thread_num = 512;
      const auto object_num = static_cast<int>(Materials.size());

      std::vector<uint> seed(1);
      std::seed_seq sequence{ std::chrono::system_clock::now().time_since_epoch().count() };
      sequence.generate( seed.begin(), seed.end() );

      cuCreatePhotonMap<<<block_num, thread_num>>>(
         Device.GlobalPhotonsPtr,
         Device.AreaLightsPtr, Device.MaterialsPtr, Device.WorldBoundsPtr, Device.ToWorldsPtr,
         Device.VertexPtr, Device.NormalPtr, Device.IndexPtr, Device.VertexSizesPtr, Device.IndexSizesPtr,
         object_num, seed[0]
      );
      CHECK_KERNEL;
      CHECK_CUDA( hipDeviceSynchronize() );
      std::cout << ">> Created Photon Map\n";

      std::cout << ">> Build Global Photon Map ...\n";
      GlobalPhotonTree = std::make_shared<KdtreeCUDA>( MaxGlobalPhotonNum, 3 );
      float* coordinates = GlobalPhotonTree->prepareDeviceCoordinatesPtr();
      cuPrepareKdtree<<<block_num, thread_num>>>( coordinates, Device.GlobalPhotonsPtr, MaxGlobalPhotonNum );
      CHECK_KERNEL;
      GlobalPhotonTree->create();
      std::cout << ">> Built Global Photon Map\n";
   }

   Mat PhotonMap::getViewMatrix(const float3& eye, const float3& center, const float3& up)
   {
      const float3 f = normalize( center - eye );
		const float3 s = normalize( cross( f, up ) );
		const float3 u = cross( s, f );

		Mat view(1.0f);
		view.c0.x = s.x;
		view.c1.x = s.y;
		view.c2.x = s.z;
		view.c0.y = u.x;
		view.c1.y = u.y;
		view.c2.y = u.z;
		view.c0.z = -f.x;
		view.c1.z = -f.y;
		view.c2.z = -f.z;
		view.c3.x = -dot( s, eye );
		view.c3.y = -dot( u, eye );
		view.c3.z = dot( f, eye );
		return view;
   }

   void PhotonMap::visualizePhotonMap(int width, int height)
   {
      std::cout << ">> Visualize Global Photon Map ...\n";
      uint8_t* image_buffer_ptr = nullptr;
      const size_t buffer_size = sizeof( uint8_t ) * width * height * 3;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&image_buffer_ptr), buffer_size ) );

      constexpr dim3 block(32, 32);
      const dim3 grid(divideUp( width, static_cast<int>(block.x) ), divideUp( height, static_cast<int>(block.y) ));
      cuVisualizePhotonMap<<<grid, block>>>(
         image_buffer_ptr,
         Device.GlobalPhotonsPtr, GlobalPhotonTree->getRoot(),
         Device.WorldBoundsPtr, Device.ToWorldsPtr,
         Device.VertexPtr, Device.NormalPtr, Device.IndexPtr, Device.VertexSizesPtr, Device.IndexSizesPtr,
         InverseViewMatrix, GlobalPhotonTree->getRootNode(), width, height, ObjectNum, MaxGlobalPhotonNum
      );
      CHECK_KERNEL;
      CHECK_CUDA( hipDeviceSynchronize() );

      auto* image_buffer = new uint8_t[width * height * 3];
      CHECK_CUDA( hipMemcpy( image_buffer, image_buffer_ptr, buffer_size, hipMemcpyDeviceToHost ) );
      hipFree( image_buffer_ptr );

      FIBITMAP* image = FreeImage_ConvertFromRawBits(
         image_buffer, width, height, width * 3, 24,
         FI_RGBA_RED_MASK, FI_RGBA_GREEN_MASK, FI_RGBA_BLUE_MASK, false
      );
      FreeImage_Save( FIF_PNG, image, "../global_photons.png" );
      FreeImage_Unload( image );
      delete [] image_buffer;
      std::cout << ">> Visualized Global Photon Map\n";
   }

   void PhotonMap::render(int width, int height)
   {
      std::cout << ">> Rendering ...\n";
      uint8_t* image_buffer_ptr = nullptr;
      const size_t buffer_size = sizeof( uint8_t ) * width * height * 3;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&image_buffer_ptr), buffer_size ) );

      std::vector<uint> seed(1);
      std::seed_seq sequence{ std::chrono::system_clock::now().time_since_epoch().count() };
      sequence.generate( seed.begin(), seed.end() );

      constexpr dim3 block(16, 16);
      const dim3 grid(divideUp( width, static_cast<int>(block.x) ), divideUp( height, static_cast<int>(block.y) ));
      cuRenderScene<<<grid, block>>>(
         image_buffer_ptr,
         Device.GlobalPhotonsPtr, GlobalPhotonTree->getRoot(),
         Device.AreaLightsPtr, Device.MaterialsPtr, Device.WorldBoundsPtr, Device.ToWorldsPtr,
         Device.VertexPtr, Device.NormalPtr, Device.IndexPtr, Device.VertexSizesPtr, Device.IndexSizesPtr,
         InverseViewMatrix, GlobalPhotonTree->getRootNode(), width, height, ObjectNum, MaxGlobalPhotonNum, seed[0]
      );
      CHECK_KERNEL;

      auto* image_buffer = new uint8_t[width * height * 3];
      CHECK_CUDA( hipMemcpy( image_buffer, image_buffer_ptr, buffer_size, hipMemcpyDeviceToHost ) );
      hipFree( image_buffer_ptr );

      FIBITMAP* image = FreeImage_ConvertFromRawBits(
         image_buffer, width, height, width * 3, 24,
         FI_RGBA_RED_MASK, FI_RGBA_GREEN_MASK, FI_RGBA_BLUE_MASK, false
      );
      FreeImage_Save( FIF_PNG, image, "../scene.png" );
      FreeImage_Unload( image );
      delete [] image_buffer;
      std::cout << ">> Rendered\n";
   }

   void PhotonMap::findNormals(
      std::vector<float3>& normals,
      const std::vector<float3>& vertices,
      const std::vector<int>& vertex_indices
   )
   {
      normals.resize( vertices.size() );
      const auto size = static_cast<int>(vertex_indices.size());
      for (int i = 0; i < size; i += 3) {
         const int n0 = vertex_indices[i];
         const int n1 = vertex_indices[i + 1];
         const int n2 = vertex_indices[i + 2];
         const float3 normal = cross( vertices[n1] - vertices[n0], vertices[n2] - vertices[n0] );
         normals[n0] += normal;
         normals[n1] += normal;
         normals[n2] += normal;
      }
      for (auto& n : normals) n = normalize( n );
   }

   void PhotonMap::readObjectFile(Box& box, const Mat& t, const std::string& file_path)
   {
      std::ifstream file(file_path);

      assert( file.is_open() );

      constexpr auto m = std::numeric_limits<float>::max();
      constexpr auto n = std::numeric_limits<float>::lowest();
      box.MinPoint = make_float3( m, m, m );
      box.MaxPoint = make_float3( n, n, n );

      bool found_normals = false;
      std::vector<float3> vertex_buffer, normal_buffer;
      std::vector<int> vertex_indices, normal_indices;
      while (!file.eof()) {
         std::string word;
         file >> word;

         if (word == "v") {
            float3 vertex;
            file >> vertex.x >> vertex.y >> vertex.z;
            vertex_buffer.emplace_back( vertex );

            const float3 v = transform( t, vertex );
            box.MinPoint.x = std::min( v.x, box.MinPoint.x );
            box.MinPoint.y = std::min( v.y, box.MinPoint.y );
            box.MinPoint.z = std::min( v.z, box.MinPoint.z );
            box.MaxPoint.x = std::max( v.x, box.MaxPoint.x );
            box.MaxPoint.y = std::max( v.y, box.MaxPoint.y );
            box.MaxPoint.z = std::max( v.z, box.MaxPoint.z );
         }
         else if (word == "vn") {
            float3 normal;
            file >> normal.x >> normal.y >> normal.z;
            normal_buffer.emplace_back( normal );
            found_normals = true;
         }
         else if (word == "f") {
            std::string face;
            const std::regex delimiter("[/]");
            for (int i = 0; i < 3; ++i) {
               file >> face;
               const std::sregex_token_iterator it(face.begin(), face.end(), delimiter, -1);
               const std::vector<std::string> vtn(it, std::sregex_token_iterator());
               vertex_indices.emplace_back( std::stoi( vtn[0] ) - 1 );
               if (found_normals && isNumber( vtn[2] )) {
                  normal_indices.emplace_back( std::stoi( vtn[2] ) - 1 );
                  found_normals = false;
               }
            }
         }
         else std::getline( file, word );
      }

      if (!found_normals) findNormals( normal_buffer, vertex_buffer, vertex_indices );

      VertexSizes.emplace_back( vertex_buffer.size() );
      IndexSizes.emplace_back( vertex_indices.size() );
      Vertices.insert(
         Vertices.end(),
         std::make_move_iterator( vertex_buffer.begin() ),
         std::make_move_iterator( vertex_buffer.end() )
      );
      Normals.insert(
         Normals.end(),
         std::make_move_iterator( normal_buffer.begin() ),
         std::make_move_iterator( normal_buffer.end() )
      );
      Indices.insert(
         Indices.end(),
         std::make_move_iterator( vertex_indices.begin() ),
         std::make_move_iterator( vertex_indices.end() )
      );
   }

   Material PhotonMap::getMaterial(const std::string& mtl_file_path)
   {
      std::ifstream file(mtl_file_path);

      assert( file.is_open() );

      Material material;
      while (!file.eof()) {
         std::string line;
         std::getline( file, line );

         const std::regex space_delimiter("[ ]");
         const std::sregex_token_iterator line_it(line.begin(), line.end(), space_delimiter, -1);
         const std::vector<std::string> parsed(line_it, std::sregex_token_iterator());
         if (parsed.empty()) continue;

         if (parsed[0] == "Ka") {
            material.Ambient.x = std::stof( parsed[1] );
            material.Ambient.y = std::stof( parsed[2] );
            material.Ambient.z = std::stof( parsed[3] );
         }
         else if (parsed[0] == "Kd") {
            material.Diffuse.x = std::stof( parsed[1] );
            material.Diffuse.y = std::stof( parsed[2] );
            material.Diffuse.z = std::stof( parsed[3] );
         }
         else if (parsed[0] == "Ks") {
            material.Specular.x = std::stof( parsed[1] );
            material.Specular.y = std::stof( parsed[2] );
            material.Specular.z = std::stof( parsed[3] );
         }
         else if (parsed[0] == "Ke") {
            material.Emission.x = std::stof( parsed[1] );
            material.Emission.y = std::stof( parsed[2] );
            material.Emission.z = std::stof( parsed[3] );
         }
         else if (parsed[0] == "Ns") material.SpecularExponent = std::stof( parsed[1] );
         else if (parsed[0] == "Ni") material.RefractiveIndex = std::stof( parsed[1] );
         else if (parsed[0] == "illum") {
            switch (std::stoi( parsed[1] )) {
               case 5: material.MaterialType = MATERIAL_TYPE::MIRROR; break;
               case 7: material.MaterialType = MATERIAL_TYPE::GLASS; break;
               default: material.MaterialType = MATERIAL_TYPE::LAMBERT; break;
            }
         }
      }
      return material;
   }

   void PhotonMap::setObjects(const std::vector<std::tuple<std::string, std::string, cuda::Mat>>& objects)
   {
      Box box;
      for (size_t i = 0; i < objects.size(); ++i) {
         Materials.emplace_back( getMaterial( std::get<1>( objects[i] ) ) );
         ToWorlds.emplace_back( std::get<2>( objects[i] ) );
         readObjectFile( box, ToWorlds[i], std::get<0>( objects[i] ) );
         WorldBounds.emplace_back( box.MinPoint, box.MaxPoint );
      }
   }

   void PhotonMap::setLights(const std::vector<std::tuple<std::string, std::string, cuda::Mat>>& lights)
   {
      Box box;
      for (const auto& light : lights) {
         const auto offset = static_cast<int>(Vertices.size());
         const auto index_offset = static_cast<int>(Indices.size());
         Materials.emplace_back( getMaterial( std::get<1>( light ) ) );
         ToWorlds.emplace_back( std::get<2>( light ) );
         readObjectFile( box, ToWorlds.back(), std::get<0>( light ) );
         WorldBounds.emplace_back( box.MinPoint, box.MaxPoint );

         const auto& m = ToWorlds.back();
         for (int j = index_offset; j < static_cast<int>(Indices.size()); j += 3) {
            const int n0 = offset + Indices[j];
            const int n1 = offset + Indices[j + 1];
            const int n2 = offset + Indices[j + 2];
            const float3 normal = cross( Vertices[n1] - Vertices[n0], Vertices[n2] - Vertices[n0] );
            AreaLights.emplace_back(
               length( normal ) * 0.5f,
               Materials.back().Emission,
               Normals[n0],
               Vertices[n0], Vertices[n1], Vertices[n2],
               m
            );
         }
      }
   }
}
#endif