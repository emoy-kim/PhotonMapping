#include "hip/hip_runtime.h"
#include "cuda/photon_map.cuh"

#ifdef USE_CUDA
namespace cuda
{
   __host__ __device__ __forceinline__
   Mat inverse(const Mat& m)
   {
      const float coef00 = m.c2.z * m.c3.w - m.c3.z * m.c2.w;
      const float coef02 = m.c1.z * m.c3.w - m.c3.z * m.c1.w;
      const float coef03 = m.c1.z * m.c2.w - m.c2.z * m.c1.w;

      const float coef04 = m.c2.y * m.c3.w - m.c3.y * m.c2.w;
      const float coef06 = m.c1.y * m.c3.w - m.c3.y * m.c1.w;
      const float coef07 = m.c1.y * m.c2.w - m.c2.y * m.c1.w;

      const float coef08 = m.c2.y * m.c3.z - m.c3.y * m.c2.z;
      const float coef10 = m.c1.y * m.c3.z - m.c3.y * m.c1.z;
      const float coef11 = m.c1.y * m.c2.z - m.c2.y * m.c1.z;

      const float coef12 = m.c2.x * m.c3.w - m.c3.x * m.c2.w;
      const float coef14 = m.c1.x * m.c3.w - m.c3.x * m.c1.w;
      const float coef15 = m.c1.x * m.c2.w - m.c2.x * m.c1.w;

      const float coef16 = m.c2.x * m.c3.z - m.c3.x * m.c2.z;
      const float coef18 = m.c1.x * m.c3.z - m.c3.x * m.c1.z;
      const float coef19 = m.c1.x * m.c2.z - m.c2.x * m.c1.z;

      const float coef20 = m.c2.x * m.c3.y - m.c3.x * m.c2.y;
      const float coef22 = m.c1.x * m.c3.y - m.c3.x * m.c1.y;
      const float coef23 = m.c1.x * m.c2.y - m.c2.x * m.c1.y;

      const float4 fac0 = make_float4( coef00, coef00, coef02, coef03 );
      const float4 fac1 = make_float4( coef04, coef04, coef06, coef07 );
      const float4 fac2 = make_float4( coef08, coef08, coef10, coef11 );
      const float4 fac3 = make_float4( coef12, coef12, coef14, coef15 );
      const float4 fac4 = make_float4( coef16, coef16, coef18, coef19 );
      const float4 fac5 = make_float4( coef20, coef20, coef22, coef23 );

      const float4 vec0 = make_float4( m.c1.x, m.c0.x, m.c0.x, m.c0.x );
      const float4 vec1 = make_float4( m.c1.y, m.c0.y, m.c0.y, m.c0.y );
      const float4 vec2 = make_float4( m.c1.z, m.c0.z, m.c0.z, m.c0.z );
      const float4 vec3 = make_float4( m.c1.w, m.c0.w, m.c0.w, m.c0.w );

      const float4 inv0 = vec1 * fac0 - vec2 * fac1 + vec3 * fac2;
      const float4 inv1 = vec0 * fac0 - vec2 * fac3 + vec3 * fac4;
      const float4 inv2 = vec0 * fac1 - vec1 * fac3 + vec3 * fac5;
      const float4 inv3 = vec0 * fac2 - vec1 * fac4 + vec2 * fac5;

      const float4 sign_a = make_float4( 1.0f, -1.0f, 1.0f, -1.0f );
      const float4 sign_b = make_float4( -1.0f, 1.0f, -1.0f, 1.0f );
      Mat inv(inv0 * sign_a, inv1 * sign_b, inv2 * sign_a, inv3 * sign_b);
      const float one_over_det = 1.0f / dot( m.c0, make_float4( inv.c0.x, inv.c1.x, inv.c2.x, inv.c3.x ) );
      inv.c0 *= one_over_det;
      inv.c1 *= one_over_det;
      inv.c2 *= one_over_det;
      inv.c3 *= one_over_det;
      return inv;
   }

   __host__ __device__ __forceinline__
   Mat transpose(const Mat& m)
   {
      Mat t;
      t.c0.x = m.c0.x;
      t.c0.y = m.c1.x;
      t.c0.z = m.c2.x;
      t.c0.w = m.c3.x;

      t.c1.x = m.c0.y;
      t.c1.y = m.c1.y;
      t.c1.z = m.c2.y;
      t.c1.w = m.c3.y;

      t.c2.x = m.c0.z;
      t.c2.y = m.c1.z;
      t.c2.z = m.c2.z;
      t.c2.w = m.c3.z;

      t.c3.x = m.c0.w;
      t.c3.y = m.c1.w;
      t.c3.z = m.c2.w;
      t.c3.w = m.c3.w;
      return t;
   }

   __host__ __device__ __forceinline__
   float3 transform(const Mat& m, const float3& v)
   {
      return make_float3(
         m.c0.x * v.x + m.c1.x * v.y + m.c2.x * v.z + m.c3.x,
         m.c0.y * v.x + m.c1.y * v.y + m.c2.y * v.z + m.c3.y,
         m.c0.z * v.x + m.c1.z * v.y + m.c2.z * v.z + m.c3.z
      );
   }

   __host__ __device__ __forceinline__
   Mat getVectorTransform(const Mat& m)
   {
      return transpose( inverse( m ) );
   }

   __host__ __device__ __forceinline__
   float3 transformVector(const Mat& m, const float3& v)
   {
      return make_float3(
         m.c0.x * v.x + m.c1.x * v.y + m.c2.x * v.z,
         m.c0.y * v.x + m.c1.y * v.y + m.c2.y * v.z,
         m.c0.z * v.x + m.c1.z * v.y + m.c2.z * v.z
      );
   }

   __device__ __forceinline__
   float getRandomValue(hiprandState* state, float a, float b)
   {
      float r = hiprand_uniform( state );
      r = r * (b - a) + a;
      return r;
   }

   // this hemisphere is towards the y-axis, and its lower plane is the xz-plane.
   __device__
   float3 getRandomPointInUnitHemisphere(hiprandState* state)
   {
      const float phi = 2.0f * HIP_PI_F * getRandomValue( state, 0.0f, 1.0f ); // [0, 2pi]
      const float theta = acos( getRandomValue( state, -1.0f, 1.0f ) ) * 0.5f; // [0, pi/2]
      const float cos_theta = cos( theta );
      return { cos( phi ) * sin( theta ), cos_theta, sin( phi ) * sin( theta ) };
   }

   __device__
   float3 getSamplePointAroundAxis(const float3& v, hiprandState* state)
   {
      const float3 u = abs( v.y ) < 0.9f ?
         normalize( cross( v, make_float3( 0.0f, 1.0f, 0.0f ) ) ) :
         normalize( cross( v, make_float3( 0.0f, 0.0f, 1.0f ) ) );
      const float3 n = normalize( cross( u, v ) );
      const float3 p = getRandomPointInUnitHemisphere( state );
      return make_float3(
         u.x * p.x + v.x * p.y + n.x * p.z,
         u.y * p.x + v.y * p.y + n.y * p.z,
         u.z * p.x + v.z * p.y + n.z * p.z
      );
   }

   __device__
   float3 getSampleRayFromLight(
      float3& ray_origin,
      float3& ray_direction,
      hiprandState* state,
      const AreaLight* lights,
      int light_num
   )
   {
      const auto light_index =
         static_cast<int>(round( getRandomValue( state, 0.0f, static_cast<float>(light_num - 1) ) ));
      const float3 v0 = lights[light_index].Vertex0;
      const float3 v1 = lights[light_index].Vertex1;
      const float3 v2 = lights[light_index].Vertex2;
      const float a = getRandomValue( state, 0.0f, 1.0f );
      const float b = getRandomValue( state, 0.0f, 1.0f );
      ray_origin = (1.0f - a - b) * v0 + a * v1 + b * v2;
      ray_origin = transform( lights[light_index].ToWorld, ray_origin );

      const float3 normal = lights[light_index].Normal;
      ray_direction = getSamplePointAroundAxis( normal, state );
      const Mat n = getVectorTransform( lights[light_index].ToWorld );
      ray_direction = normalize( transformVector( n, ray_direction ) );
      ray_origin += RayEpsilon * ray_direction;
      return lights[light_index].Color;
   }

   __device__
   bool intersectWithBox(const float3& ray_origin, const float3& ray_direction, const Box& box)
   {
      float exit = HIP_INF_F;
      float enter = -HIP_INF_F;
      if (abs( ray_direction.x ) > 1e-5f) {
         const float t_min = (box.MinPoint.x - ray_origin.x) / ray_direction.x;
         const float t_max = (box.MaxPoint.x - ray_origin.x) / ray_direction.x;
         const float t_enter = min( t_min, t_max );
         const float t_exit = max( t_min, t_max );
         if (t_enter > enter) enter = t_enter;
         if (t_exit < exit) exit = t_exit;
         if (enter > exit || exit < 0.0f) return false;
      }
      else if (ray_origin.x < box.MinPoint.x || box.MaxPoint.x < ray_origin.x) return false;

      if (abs( ray_direction.y ) > 1e-5f) {
         const float t_min = (box.MinPoint.y - ray_origin.y) / ray_direction.y;
         const float t_max = (box.MaxPoint.y - ray_origin.y) / ray_direction.y;
         const float t_enter = min( t_min, t_max );
         const float t_exit = max( t_min, t_max );
         if (t_enter > enter) enter = t_enter;
         if (t_exit < exit) exit = t_exit;
         if (enter > exit || exit < 0.0f) return false;
      }
      else if (ray_origin.y < box.MinPoint.y || box.MaxPoint.y < ray_origin.y) return false;

      if (abs( ray_direction.z ) > 1e-5f) {
         const float t_min = (box.MinPoint.z - ray_origin.z) / ray_direction.z;
         const float t_max = (box.MaxPoint.z - ray_origin.z) / ray_direction.z;
         const float t_enter = min( t_min, t_max );
         const float t_exit = max( t_min, t_max );
         if (t_enter > enter) enter = t_enter;
         if (t_exit < exit) exit = t_exit;
         if (enter > exit || exit < 0.0f) return false;
      }
      else if (ray_origin.z < box.MinPoint.z || box.MaxPoint.z < ray_origin.z) return false;

      //distance = enter > 0.0f ? enter : exit;
      return true;
   }

   __device__
   bool intersectWithTriangle(
      float3& tuv,
      const float3& ray_origin,
      const float3& ray_direction,
      const float3& p0,
      const float3& p1,
      const float3& p2
   )
   {
      const float3 e1 = p1 - p0;
      const float3 e2 = p2 - p0;
      const float3 q = cross( ray_direction, e2 );
      const float det = dot( e1, q );
      if (abs( det ) < 1e-5f) return false;

      const float f = 1.0f / det;
      const float3 s = ray_origin - p0;
      const float u = f * dot( s, q );
      if (u < 0.0f) return false;

      const float3 r = cross( s, e1 );
      const float v = f * dot( ray_direction, r );
      if (v < 0.0f || u + v > 1.0f) return false;

      const float t = f * dot( e2, r );
      if (t <= 0.0f) return false;

      tuv = make_float3( t, u, v );
      return true;
   }

   __device__
   bool findIntersection(
      IntersectionInfo& intersection,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      const float3& ray_origin,
      const float3& ray_direction,
      int object_num
   )
   {
      bool intersect = false;
      float distance = HIP_INF_F;
      int offset = 0, index_offset = 0;
      for (int i = 0; i < object_num; ++i) {
         if (intersectWithBox( ray_origin, ray_direction, world_bounds[i] )) {
            const Mat vector_transform = getVectorTransform( to_worlds[i] );
            for (int j = 0; j < index_sizes[i]; j += 3) {
               const int k0 = offset + indices[index_offset + j];
               const int k1 = offset + indices[index_offset + j + 1];
               const int k2 = offset + indices[index_offset + j + 2];
               const float3 p0 = transform( to_worlds[i], vertices[k0] );
               const float3 p1 = transform( to_worlds[i], vertices[k1] );
               const float3 p2 = transform( to_worlds[i], vertices[k2] );

               float3 tuv;
               if (intersectWithTriangle( tuv, ray_origin, ray_direction, p0, p1, p2 )) {
                  if (distance > tuv.x) {
                     const float3 n0 = normalize( transformVector( vector_transform, normals[k0] ) );
                     const float3 n1 = normalize( transformVector( vector_transform, normals[k1] ) );
                     const float3 n2 = normalize( transformVector( vector_transform, normals[k2] ) );
                     distance = tuv.x;
                     intersection.ObjectIndex = i;
                     intersection.Position = ray_origin + tuv.x * ray_direction;
                     intersection.ShadingNormal = (1.0f - tuv.y - tuv.z) * n0 + tuv.y * n1 + tuv.z * n2;
                     intersect = true;
                  }
               }
            }
         }
         offset += vertex_sizes[i];
         index_offset += index_sizes[i];
      }
      return intersect;
   }

   __device__
   float getSchlickApproximation(float cos_theta, float n1, float n2)
   {
      const float r0 = (n1 - n2) * (n1 - n2) / ((n1 + n2) * (n1 + n2));
      const float x = 1.0f - abs( cos_theta );
      const float x2 = x * x;
      return r0 + (1.0f - r0) * x2 * x2 * x;
   }

   __device__
   float3 getRefractiveDirection(
      const float3& ray_direction,
      const float3& normal,
      float in_refractive_index,
      float out_refractive_index
   )
   {
      float3 n;
      float eta;
      if (dot( -ray_direction, normal ) >= 0.0f) {
         n = normal;
         eta = in_refractive_index / out_refractive_index;
      }
      else {
         n = -normal;
         eta = out_refractive_index / in_refractive_index;
      }

      float3 outgoing = refract( ray_direction, n, eta );
      if (outgoing.x == 0.0f && outgoing.y == 0.0f && outgoing.z == 0.0f) {
         outgoing = reflect( ray_direction, n );
      }
      return outgoing;
   }

   __device__ int emitted_photon_num;

   __device__ __forceinline__ float max3(float3 a)
   {
      return max( a.x, max( a.y, a.z ) );
   }

   __device__ __forceinline__ float mean(float3 a)
   {
      return (a.x + a.y + a.z) / 3.0f;
   }

   __global__
   void cuCreatePhotonMap(
      Photon* photons,
      const AreaLight* lights,
      const Material* materials,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      int light_num,
      int object_num,
      uint seed
   )
   {
      int generated_num = 0;
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      const int photons_to_generate = divideUp( MaxGlobalPhotonNum, step );
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x) * photons_to_generate;

      hiprandState state;
      hiprand_init( seed, index, 0, &state );

      while (true) {
         float in_refractive_index = 1.0f;
         float3 ray_origin, ray_direction;
         float3 color = getSampleRayFromLight( ray_origin, ray_direction, &state, lights, light_num );
         atomicAdd( &emitted_photon_num, 1 );
         for (int i = 0; i < MaxDepth; ++i) {
            IntersectionInfo intersection;
            if (!findIntersection(
                  intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
                  ray_origin, ray_direction, object_num
               )) break;

            const Material& m = materials[intersection.ObjectIndex];
            if (m.useDiffuse()) {
               photons[index + generated_num].Power = color;
               photons[index + generated_num].Position = intersection.Position;
               photons[index + generated_num].IncomingDirection = -ray_direction;
               generated_num++;
               if (generated_num == photons_to_generate || index + generated_num >= MaxGlobalPhotonNum) return;
            }

            float fresnel = 0.0f;
            if (m.useRefractionRay()) {
               fresnel = getSchlickApproximation(
                  dot( -ray_direction, intersection.ShadingNormal ), in_refractive_index, m.RefractiveIndex
               );
            }
            const float mean_color = mean( color );
            const float diffuse_probability = mean( m.Diffuse * color / mean_color );
            const float specular_probability =
               m.useReflectionRay() ? mean( m.Specular * color / mean_color ) + fresnel : 0.0f;
            const float refractive_probability = m.useRefractionRay() ? 1.0f - fresnel : 0.0f;
            constexpr float terminate_probability = 0.005f;
            const float probability_sum =
               diffuse_probability + specular_probability + refractive_probability + terminate_probability;
            float r = getRandomValue( &state, 0.0f, 1.0f );
            if (probability_sum > 1.0f) r *= probability_sum;

            float3 outgoing;
            if (r < diffuse_probability) {
               outgoing = getSamplePointAroundAxis( intersection.ShadingNormal, &state );
               color *= m.Diffuse / diffuse_probability;
            }
            else if (r < diffuse_probability + refractive_probability) {
               outgoing = getRefractiveDirection(
                  ray_direction, intersection.ShadingNormal, in_refractive_index, m.RefractiveIndex
               );
               color *= (1.0f - fresnel) / refractive_probability;
            }
            else if (r < diffuse_probability + specular_probability + refractive_probability) {
               outgoing = reflect( ray_direction, intersection.ShadingNormal );
               color *= (m.Specular + fresnel) / specular_probability;
            }
            else break;

            ray_direction = outgoing;
            ray_origin = intersection.Position;
            in_refractive_index = m.RefractiveIndex;
         }
      }
   }

   __global__
   void cuScalePower(Photon* photons, float total_light_power, int size)
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      const auto scale = total_light_power / static_cast<float>(emitted_photon_num);
      for (int i = index; i < size; i += step) {
         photons[i].Power *= scale;
      }
   }

   __global__
   void cuPrepareKdtree(float* coordinates, const Photon* photons, int size)
   {
      const auto index = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto step = static_cast<int>(blockDim.x * gridDim.x);
      for (int i = index; i < size; i += step) {
         coordinates[3 * i] = photons[i].Position.x;
         coordinates[3 * i + 1] = photons[i].Position.y;
         coordinates[3 * i + 2] = photons[i].Position.z;
      }
   }

   __device__
   void findNearestNeighbor(
      int& found_index,
      float& found_distance,
      const KdtreeNode* root,
      const Photon* photons,
      const float3& query,
      int node_index,
      int size
   )
   {
      constexpr int dim = 3;
      int depth = 0;
      int prev = -1;
      int curr = node_index;
      found_index = -1;
      found_distance = HIP_INF_F;
      while (curr >= 0) {
         const KdtreeNode* node = &root[curr];
         const int parent = node->ParentIndex;
         if (curr >= size) {
            prev = curr;
            curr = parent;
            continue;
         }

         const bool from_child = prev >= 0 && (prev == node->LeftChildIndex || prev == node->RightChildIndex);
         if (!from_child) {
            const float3 v = query - photons[node->Index].Position;
            const float squared_distance = dot( v, v );
            if (squared_distance <= found_distance) {
               found_index = curr;
               found_distance = squared_distance;
            }
         }

         float t;
         const int axis = depth % dim;
         if (axis == 0) t = query.x - photons[node->Index].Position.x;
         else if (axis == 1) t = query.y - photons[node->Index].Position.y;
         else t = query.z - photons[node->Index].Position.z;
         const bool right_priority = t > 0;
         const int far_child = right_priority ? node->LeftChildIndex : node->RightChildIndex;
         const int close_child = right_priority ? node->RightChildIndex : node->LeftChildIndex;

         int next = -1;
         if (prev >= 0 && prev == close_child) {
            if (far_child >= 0 && (t == 0 || t * t <= found_distance)) {
               next = far_child;
               depth++;
            }
            else {
               next = parent;
               depth--;
            }
         }
         else if (prev >= 0 && prev == far_child) {
            next = parent;
            depth--;
         }
         else if (prev < 0 || prev == parent) {
            if (close_child < 0 && far_child < 0) {
               next = parent;
               depth--;
            }
            else if (close_child < 0) {
               next = far_child;
               depth++;
            }
            else {
               next = close_child;
               depth++;
            }
         }

         prev = curr;
         curr = next;
      }
   }

   __device__
   float push(int found_index[], float found_distance[], int node_index, float squared_distance)
   {
      int n = node_index;
      float d = squared_distance;
      for (int i = 0; i < NeighborNum; ++i) {
         float max_distance = found_distance[i];
         float min_distance = d;
         int max_index = found_index[i];
         int min_index = n;
         if (max_distance < min_distance) {
            max_distance = d;
            min_distance = found_distance[i];
            max_index = n;
            min_index = found_index[i];
         }
         found_index[i] = min_index;
         found_distance[i] = min_distance;
         n = max_index;
         d = max_distance;
      }
      return found_distance[NeighborNum - 1];
   }

   __device__
   void findNearestNeighbors(
      int found_index[],
      float found_distance[],
      const KdtreeNode* root,
      const Photon* photons,
      const float3& query,
      int node_index,
      int size
   )
   {
      constexpr int dim = 3;
      int depth = 0;
      int prev = -1;
      int curr = node_index;
      float max_squared_distance = HIP_INF_F;
      while (curr >= 0) {
         const KdtreeNode* node = &root[curr];
         const int parent = node->ParentIndex;
         if (curr >= size) {
            prev = curr;
            curr = parent;
            continue;
         }

         const bool from_child = prev >= 0 && (prev == node->LeftChildIndex || prev == node->RightChildIndex);
         if (!from_child) {
            const float3 v = query - photons[node->Index].Position;
            const float squared_distance = dot( v, v );
            if (squared_distance <= max_squared_distance) {
               max_squared_distance = push( found_index, found_distance, curr, squared_distance );
            }
         }

         float t;
         const int axis = depth % dim;
         if (axis == 0) t = query.x - photons[node->Index].Position.x;
         else if (axis == 1) t = query.y - photons[node->Index].Position.y;
         else t = query.z - photons[node->Index].Position.z;
         const bool right_priority = t > 0;
         const int far_child = right_priority ? node->LeftChildIndex : node->RightChildIndex;
         const int close_child = right_priority ? node->RightChildIndex : node->LeftChildIndex;

         int next = -1;
         if (prev >= 0 && prev == close_child) {
            if (far_child >= 0 && (t == 0 || t * t <= max_squared_distance)) {
               next = far_child;
               depth++;
            }
            else {
               next = parent;
               depth--;
            }
         }
         else if (prev >= 0 && prev == far_child) {
            next = parent;
            depth--;
         }
         else if (prev < 0 || prev == parent) {
            if (close_child < 0 && far_child < 0) {
               next = parent;
               depth--;
            }
            else if (close_child < 0) {
               next = far_child;
               depth++;
            }
            else {
               next = close_child;
               depth++;
            }
         }

         prev = curr;
         curr = next;
      }
   }

   __global__
   void cuVisualizePhotonMap(
      uint8_t* image_buffer,
      const Photon* photons,
      const KdtreeNode* root,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      Mat inverse_view,
      int root_node,
      int width,
      int height,
      int object_num,
      int size
   )
   {
      const auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);
      if (x >= width || y >= height) return;

      const int k = (y * width + x) * 3;
      const auto w = static_cast<float>(width);
      const auto h = static_cast<float>(height);
      const float u = (2.0f * static_cast<float>(x) - w) / h;
      const float v = (2.0f * static_cast<float>(y) - h) / h;
      const float3 ray_origin = make_float3( inverse_view.c3.x, inverse_view.c3.y, inverse_view.c3.z );
      const float3 ray_direction = normalize( transform( inverse_view, make_float3( u, v, -1.0f ) ) - ray_origin );

      IntersectionInfo intersection;
      if (!findIntersection(
            intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
            ray_origin, ray_direction, object_num
         )) {
         image_buffer[k] = image_buffer[k + 1] = image_buffer[k + 2] = 0;
         return;
      }

      int index;
      float squared_distance;
      findNearestNeighbor( index, squared_distance, root, photons, intersection.Position, root_node, size );
      if (squared_distance < 0.1f) {
         float3 power = photons[index].Power * static_cast<float>(emitted_photon_num);
         power.x = min( max( power.x * 255.0f, 0.0f ), 255.0f );
         power.y = min( max( power.y * 255.0f, 0.0f ), 255.0f );
         power.z = min( max( power.z * 255.0f, 0.0f ), 255.0f );
         image_buffer[k + 2] = static_cast<uint8_t>(power.x);
         image_buffer[k + 1] = static_cast<uint8_t>(power.y);
         image_buffer[k] = static_cast<uint8_t>(power.z);
      }
      else image_buffer[k] = image_buffer[k + 1] = image_buffer[k + 2] = 0;
   }

   __device__
   bool hitLight(
      float3& emission,
      const float3& ray_origin,
      const float3& ray_direction,
      const AreaLight* lights,
      int light_num
   )
   {
      for (int i = 0; i < light_num; ++i) {
         float3 tuv;
         const float3 v0 = lights[i].Vertex0;
         const float3 v1 = lights[i].Vertex1;
         const float3 v2 = lights[i].Vertex2;
         const float3 p0 = transform( lights[i].ToWorld, v0 );
         const float3 p1 = transform( lights[i].ToWorld, v1 );
         const float3 p2 = transform( lights[i].ToWorld, v2 );
         if (intersectWithTriangle( tuv, ray_origin, ray_direction, p0, p1, p2 )) {
            emission = lights[i].Emission;
            return true;
         }
      }
      return false;
   }

   __device__
   float3 computeRadianceWithPhotonMap(
      const IntersectionInfo& intersection,
      const KdtreeNode* root,
      const Photon* photons,
      const Material* materials,
      const float3& ray_direction,
      int root_node,
      int size
   )
   {
      int indices[NeighborNum];
      float squared_distances[NeighborNum];
      for (int i = 0; i < NeighborNum; ++i) {
         indices[i] = -1;
         squared_distances[i] = HIP_INF_F;
      }
      findNearestNeighbors( indices, squared_distances, root, photons, intersection.Position, root_node, size );

      int k = -1;
      for (int i = NeighborNum - 1; i >= 0; --i) {
         if (indices[i] >= 0) {
            k = i;
            break;
         }
      }

      float3 radiance = make_float3( 0.0f, 0.0f, 0.0f );
      const Material& m = materials[intersection.ObjectIndex];
      const float max_distance = k < 0 ? 0.0f : squared_distances[k];
      const float v_dot_n = dot( -ray_direction, intersection.ShadingNormal );
      const float3 outgoing = reflect( ray_direction, intersection.ShadingNormal );
      for (int i = 0; i < k; ++i) {
         const float n = dot( photons[indices[i]].IncomingDirection, intersection.ShadingNormal );
         if (v_dot_n * n < 0.0f) continue;

         float3 color = make_float3( 0.0f, 0.0f, 0.0f );
         if (m.useDiffuse()) color += abs( n ) * m.Diffuse;
         if (m.useSpecular()) {
            const float d = max( dot( outgoing, photons[indices[i]].IncomingDirection ), 0.0f );
            color += pow( d, m.SpecularExponent ) * m.Specular;
         }
         radiance += photons[indices[i]].Power * color;
      }
      if (max_distance >= 0.0f) radiance /= HIP_PI_F * max_distance;
      return radiance;
   }

   __device__
   float3 computeDirectIllumination(
      const IntersectionInfo& intersection,
      const AreaLight* lights,
      const Material* materials,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3& ray_direction,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      hiprandState* state,
      int light_num,
      int object_num
   )
   {
      float3 radiance = make_float3( 0.0f, 0.0f, 0.0f );
      const Material& m = materials[intersection.ObjectIndex];
      const float v_dot_n = dot( -ray_direction, intersection.ShadingNormal );
      for (int i = 0; i < light_num; ++i) {
         const float3 v0 = lights[i].Vertex0;
         const float3 v1 = lights[i].Vertex1;
         const float3 v2 = lights[i].Vertex2;
         const float a = getRandomValue( state, 0.0f, 1.0f );
         const float b = getRandomValue( state, 0.0f, 1.0f );
         float3 light_position = (1.0f - a - b) * v0 + a * v1 + b * v2;
         light_position = transform( lights[i].ToWorld, light_position );

         const float3 l = normalize( light_position - intersection.Position );
         const float diffuse_intensity = dot( l, intersection.ShadingNormal );
         if (v_dot_n * diffuse_intensity < 0.0f) continue;

         IntersectionInfo shadow_intersection;
         const float3 origin = intersection.Position + RayEpsilon * l;
         const float length_to_light = length( light_position - origin );
         const bool intersect = findIntersection(
            shadow_intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
            origin, l, object_num
         );
         if (!intersect || length( shadow_intersection.Position - origin ) >= length_to_light - RayEpsilon) {
            const float3 light_normal =
               normalize( transformVector( getVectorTransform( lights[i].ToWorld ), lights[i].Normal ) );
            const float3 light_intensity = lights[i].Color * max( dot( -l, light_normal ), 0.0f );
            if (m.useDiffuse()) radiance += light_intensity * abs( diffuse_intensity ) * m.Diffuse;
            if (m.useSpecular()) {
               const float specular_intensity = dot( -ray_direction, reflect( -l, intersection.ShadingNormal ) );
               if (specular_intensity > 0.0f) {
                  radiance += light_intensity * m.Specular * pow( specular_intensity, m.SpecularExponent );
               }
            }
            if (m.useAmbient()) radiance += lights[i].Ambient * m.Ambient;
         }
      }
      return radiance + m.Emission;
   }

   __device__
   float3 computeIndirectIllumination(
      const IntersectionInfo& intersection,
      const KdtreeNode* root,
      const Photon* photons,
      const AreaLight* lights,
      const Material* materials,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      hiprandState* state,
      int root_node,
      int light_num,
      int object_num,
      int size
   )
   {
      float3 radiance = make_float3( 0.0f, 0.0f, 0.0f );
      float3 ray_direction = getSamplePointAroundAxis( intersection.ShadingNormal, state );
      float3 ray_origin = intersection.Position + RayEpsilon * ray_direction;
      float in_refractive_index = materials[intersection.ObjectIndex].RefractiveIndex;
      for (int i = 0; i < MaxDepth; ++i) {
         IntersectionInfo next_intersection;
         if (!findIntersection(
            next_intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
            ray_origin, ray_direction, object_num
         )) break;

         float3 light_emission = make_float3( 0.0f, 0.0f, 0.0f );
         if (hitLight( light_emission, ray_origin, ray_direction, lights, light_num )) radiance += light_emission;

         float fresnel = 0.0f;
         const Material& m = materials[next_intersection.ObjectIndex];
         if (m.useRefractionRay()) {
            fresnel = getSchlickApproximation(
               dot( -ray_direction, next_intersection.ShadingNormal ), in_refractive_index, m.RefractiveIndex
            );
         }

         const float diffuse_probability = mean( m.Diffuse );
         const float specular_probability = m.useReflectionRay() ? mean( m.Specular ) + fresnel : 0.0f;
         const float refractive_probability = m.useRefractionRay() ? 1.0f - fresnel : 0.0f;
         constexpr float terminate_probability = 0.005f;
         const float probability_sum =
            diffuse_probability + specular_probability + refractive_probability + terminate_probability;
         float r = getRandomValue( state, 0.0f, 1.0f );
         if (probability_sum > 1.0f) r *= probability_sum;

         if (r < diffuse_probability) {
            radiance += computeRadianceWithPhotonMap(
               next_intersection, root, photons, materials, ray_direction, root_node, size
            );
            break;
         }
         else if (r < diffuse_probability + refractive_probability) {
            ray_direction = getRefractiveDirection(
               ray_direction, next_intersection.ShadingNormal, in_refractive_index, m.RefractiveIndex
            );
         }
         else if (r < diffuse_probability + specular_probability + refractive_probability) {
            ray_direction = reflect( ray_direction, next_intersection.ShadingNormal );
         }
         else break;

         ray_origin = next_intersection.Position + RayEpsilon * ray_direction;
         in_refractive_index = m.RefractiveIndex;
      }
      return radiance;
   }

   __device__
   float3 trace(
      const IntersectionInfo& intersection,
      const KdtreeNode* root,
      const Photon* photons,
      const AreaLight* lights,
      const Material* materials,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3& outgoing,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      hiprandState* state,
      int root_node,
      int light_num,
      int object_num,
      int size
   )
   {
      float3 radiance = make_float3( 0.0f, 0.0f, 0.0f );
      float3 ray_direction = outgoing;
      float3 ray_origin = intersection.Position + RayEpsilon * outgoing;
      float in_refractive_index = materials[intersection.ObjectIndex].RefractiveIndex;
      for (int i = 0; i < MaxDepth; ++i) {
         IntersectionInfo next_intersection;
         if (!findIntersection(
            next_intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
            ray_origin, ray_direction, object_num
         )) break;

         float3 light_emission = make_float3( 0.0f, 0.0f, 0.0f );
         if (hitLight( light_emission, ray_origin, ray_direction, lights, light_num )) radiance += light_emission;

         const Material& m = materials[next_intersection.ObjectIndex];
         if (m.useDiffuse()) {
            radiance += computeDirectIllumination(
               next_intersection, lights, materials, world_bounds, to_worlds, ray_direction,
               vertices, normals, indices, vertex_sizes, index_sizes, state, light_num, object_num
            );
         }
         //if (m.isDiffuse()) radiance += computeCausticsWithPhotonMap() * weight;

         float fresnel = 0.0f;
         if (m.useRefractionRay()) {
            fresnel = getSchlickApproximation(
               dot( -ray_direction, next_intersection.ShadingNormal ), in_refractive_index, m.RefractiveIndex
            );
         }

         const float diffuse_probability = mean( m.Diffuse );
         const float specular_probability = m.useReflectionRay() ? mean( m.Specular ) + fresnel : 0.0f;
         const float refractive_probability = m.useRefractionRay() ? 1.0f - fresnel : 0.0f;
         constexpr float terminate_probability = 0.005f;
         const float probability_sum =
            diffuse_probability + specular_probability + refractive_probability + terminate_probability;
         float r = getRandomValue( state, 0.0f, 1.0f );
         if (probability_sum > 1.0f) r *= probability_sum;

         if (r < diffuse_probability) {
            float3 indirect = make_float3( 0.0f, 0.0f, 0.0f );
            for (int s = 0; s < IndirectSampleNum; ++s) {
               indirect += computeIndirectIllumination(
                  next_intersection, root, photons, lights, materials, world_bounds, to_worlds,
                  vertices, normals, indices, vertex_sizes, index_sizes, state, root_node, light_num, object_num, size
               );
            }
            radiance += indirect / static_cast<float>(IndirectSampleNum);
            break;
         }
         else if (r < diffuse_probability + refractive_probability) {
            ray_direction = getRefractiveDirection(
               ray_direction, next_intersection.ShadingNormal, in_refractive_index, m.RefractiveIndex
            );
         }
         else if (r < diffuse_probability + specular_probability + refractive_probability) {
            ray_direction = reflect( ray_direction, next_intersection.ShadingNormal );
         }
         else break;

         ray_origin = next_intersection.Position + RayEpsilon * ray_direction;
         in_refractive_index = m.RefractiveIndex;
      }
      return radiance;
   }

   __device__
   float3 estimateRadiance(
      const Photon* photons,
      const AreaLight* lights,
      const Material* materials,
      const KdtreeNode* root,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3& ray_origin,
      const float3& ray_direction,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      hiprandState* state,
      int root_node,
      int light_num,
      int object_num,
      int size
   )
   {
      IntersectionInfo intersection;
      float3 radiance = make_float3( 0.0f, 0.0f, 0.0f );
      if (!findIntersection(
         intersection, world_bounds, to_worlds, vertices, normals, indices, vertex_sizes, index_sizes,
         ray_origin, ray_direction, object_num
      )) return radiance;

      float3 light_emission = make_float3( 0.0f, 0.0f, 0.0f );
      if (hitLight( light_emission, ray_origin, ray_direction, lights, light_num )) radiance += light_emission;

      const Material& m = materials[intersection.ObjectIndex];
      if (m.useDiffuse()) {
         radiance += computeDirectIllumination(
            intersection, lights, materials, world_bounds, to_worlds, ray_direction,
            vertices, normals, indices, vertex_sizes, index_sizes, state, light_num, object_num
         );
      }

      float fresnel = 0.0f;
      if (m.useRefractionRay()) {
         fresnel =
            getSchlickApproximation( dot( -ray_direction, intersection.ShadingNormal ), 1.0f, m.RefractiveIndex );
         if (fresnel < 1.0f) {
            const float3 outgoing =
               getRefractiveDirection( ray_direction, intersection.ShadingNormal, 1.0f, m.RefractiveIndex );

            float3 transmissive = make_float3( 0.0f, 0.0f, 0.0f );
            for (int i = 0; i < TransmissiveSampleNum; ++i) {
               transmissive += trace(
                  intersection, root, photons, lights, materials, world_bounds, to_worlds, outgoing,
                  vertices, normals, indices, vertex_sizes, index_sizes, state, root_node, light_num, object_num, size
               );
            }
            radiance += transmissive / static_cast<float>(TransmissiveSampleNum);
         }
      }
      if (m.useReflectionRay() || fresnel > 0.0f) {
         const float3 outgoing = reflect( ray_direction, intersection.ShadingNormal );
         radiance += trace(
            intersection, root, photons, lights, materials, world_bounds, to_worlds, outgoing,
            vertices, normals, indices, vertex_sizes, index_sizes, state, root_node, light_num, object_num, size
         ) * (m.Specular + fresnel);
      }
      if (m.useDiffuse()) {
         radiance += computeIndirectIllumination(
            intersection, root, photons, lights, materials, world_bounds, to_worlds,
            vertices, normals, indices, vertex_sizes, index_sizes, state, root_node, light_num, object_num, size
         ) * m.Diffuse;
         //radiance += computeCausticsWithPhotonMap();
         radiance +=
            computeRadianceWithPhotonMap( intersection, root, photons, materials, ray_direction, root_node, size );
      }
      return radiance;
   }

   __global__
   void cuRenderScene(
      uint8_t* image_buffer,
      const Photon* photons,
      const KdtreeNode* root,
      const AreaLight* lights,
      const Material* materials,
      const Box* world_bounds,
      const Mat* to_worlds,
      const float3* vertices,
      const float3* normals,
      const int* indices,
      const int* vertex_sizes,
      const int* index_sizes,
      Mat inverse_view,
      int root_node,
      int width,
      int height,
      int light_num,
      int object_num,
      int size,
      uint seed
   )
   {
      const auto x = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
      const auto y = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);
      if (x >= width || y >= height) return;

      hiprandState state;
      hiprand_init( seed, x * 1973 + y * 9277, 0, &state );

      const auto w = static_cast<float>(width);
      const auto h = static_cast<float>(height);
      const float3 ray_origin = make_float3( inverse_view.c3.x, inverse_view.c3.y, inverse_view.c3.z );
      float3 color = make_float3( 0.0f, 0.0f, 0.0f );
      for (int s = 0; s < SampleNum; ++s) {
         const float u = (2.0f * (static_cast<float>(x) + getRandomValue( &state, 0.0f, 1.0f )) - w) / h;
         const float v = (2.0f * (static_cast<float>(y) + getRandomValue( &state, 0.0f, 1.0f )) - h) / h;
         const float3 ray_direction = normalize( transform( inverse_view, make_float3( u, v, -1.0f ) ) - ray_origin );
         color += estimateRadiance(
            photons, lights, materials, root, world_bounds, to_worlds, ray_origin, ray_direction,
            vertices, normals, indices, vertex_sizes, index_sizes, &state, root_node, light_num, object_num, size
         );
      }
      color /= static_cast<float>(SampleNum);

      const int k = (y * width + x) * 3;
      image_buffer[k + 2] = static_cast<uint8_t>(min( max( color.x * 255.0f, 0.0f ), 255.0f ));
      image_buffer[k + 1] = static_cast<uint8_t>(min( max( color.y * 255.0f, 0.0f ), 255.0f ));
      image_buffer[k] = static_cast<uint8_t>(min( max( color.z * 255.0f, 0.0f ), 255.0f ));
   }

   PhotonMap::PhotonMap() : Device(), LightNum( 0 ), ObjectNum( 0 ), TotalLightPower( 0.0f )
   {
      ViewMatrix = getViewMatrix(
         make_float3( 0.0f, 250.0f, 750.0f ),
         make_float3( 0.0f, 250.0f, 0.0f ),
         make_float3( 0.0f, 1.0f, 0.0f )
      );
      InverseViewMatrix = inverse( ViewMatrix );
   }

   PhotonMap::~PhotonMap()
   {
      if (Device.VertexPtr != nullptr) hipFree( Device.VertexPtr );
      if (Device.VertexPtr != nullptr) hipFree( Device.VertexPtr );
      if (Device.NormalPtr != nullptr) hipFree( Device.NormalPtr );
      if (Device.IndexPtr != nullptr) hipFree( Device.IndexPtr );
      if (Device.VertexSizesPtr != nullptr) hipFree( Device.VertexSizesPtr );
      if (Device.IndexSizesPtr != nullptr) hipFree( Device.IndexSizesPtr );
      if (Device.WorldBoundsPtr != nullptr) hipFree( Device.WorldBoundsPtr );
      if (Device.ToWorldsPtr != nullptr) hipFree( Device.ToWorldsPtr );
      if (Device.MaterialsPtr != nullptr) hipFree( Device.MaterialsPtr );
      if (Device.AreaLightsPtr != nullptr) hipFree( Device.AreaLightsPtr );
   }

   void PhotonMap::initialize()
   {
      assert( Vertices.size() == Normals.size() );

      ObjectNum = static_cast<int>(Materials.size());

      int device_num = 0;
      CHECK_CUDA( hipGetDeviceCount( &device_num ) );
      if( device_num <= 0 ) throw std::runtime_error( "cuda device not found\n" );

      Device.ID = 0;
      CHECK_CUDA( hipSetDevice( Device.ID ) );

      auto buffer_size = sizeof( float3 ) * Vertices.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.VertexPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.VertexPtr, Vertices.data(), buffer_size, hipMemcpyHostToDevice ) );

      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.NormalPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.NormalPtr, Normals.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( int ) * Indices.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.IndexPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.IndexPtr, Indices.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( int ) * VertexSizes.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.VertexSizesPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.VertexSizesPtr, VertexSizes.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( int ) * IndexSizes.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.IndexSizesPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.IndexSizesPtr, IndexSizes.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( Box ) * WorldBounds.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.WorldBoundsPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.WorldBoundsPtr, WorldBounds.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( Mat ) * ToWorlds.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.ToWorldsPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.ToWorldsPtr, ToWorlds.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( Material ) * Materials.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.MaterialsPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.MaterialsPtr, Materials.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( AreaLight ) * AreaLights.size();
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.AreaLightsPtr), buffer_size ) );
      CHECK_CUDA( hipMemcpy( Device.AreaLightsPtr, AreaLights.data(), buffer_size, hipMemcpyHostToDevice ) );

      buffer_size = sizeof( Photon ) * MaxGlobalPhotonNum;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&Device.GlobalPhotonsPtr), buffer_size ) );
   }

   void PhotonMap::createPhotonMap()
   {
      initialize();

      std::cout << ">> Create Photon Map ...\n";
      constexpr int block_num = 128;
      constexpr int thread_num = 512;

      std::vector<uint> seed(1);
      std::seed_seq sequence{ std::chrono::system_clock::now().time_since_epoch().count() };
      sequence.generate( seed.begin(), seed.end() );

      int emitted_num = 0;
      CHECK_CUDA( hipMemcpyToSymbol(HIP_SYMBOL( emitted_photon_num), &emitted_num, sizeof( int ), 0, hipMemcpyHostToDevice ) );

      cuCreatePhotonMap<<<block_num, thread_num>>>(
         Device.GlobalPhotonsPtr,
         Device.AreaLightsPtr, Device.MaterialsPtr, Device.WorldBoundsPtr, Device.ToWorldsPtr,
         Device.VertexPtr, Device.NormalPtr, Device.IndexPtr, Device.VertexSizesPtr, Device.IndexSizesPtr,
         LightNum, ObjectNum, seed[0]
      );
      CHECK_KERNEL;
      CHECK_CUDA( hipDeviceSynchronize() );

      CHECK_CUDA( hipMemcpyFromSymbol( &emitted_num, HIP_SYMBOL(emitted_photon_num), sizeof( int ), 0, hipMemcpyDeviceToHost ) );
      std::cout << ">> Emitted Photon Num: " << emitted_num << "\n";

      cuScalePower<<<block_num, thread_num>>>( Device.GlobalPhotonsPtr, TotalLightPower, MaxGlobalPhotonNum );
      CHECK_KERNEL;
      std::cout << ">> Created Photon Map\n";

      std::cout << ">> Build Global Photon Map ...\n";
      GlobalPhotonTree = std::make_shared<KdtreeCUDA>( MaxGlobalPhotonNum, 3 );
      float* coordinates = GlobalPhotonTree->prepareDeviceCoordinatesPtr();
      cuPrepareKdtree<<<block_num, thread_num>>>( coordinates, Device.GlobalPhotonsPtr, MaxGlobalPhotonNum );
      CHECK_KERNEL;
      GlobalPhotonTree->create();
      std::cout << ">> Built Global Photon Map\n";
   }

   Mat PhotonMap::getViewMatrix(const float3& eye, const float3& center, const float3& up)
   {
      const float3 f = normalize( center - eye );
		const float3 s = normalize( cross( f, up ) );
		const float3 u = cross( s, f );

		Mat view(1.0f);
		view.c0.x = s.x;
		view.c1.x = s.y;
		view.c2.x = s.z;
		view.c0.y = u.x;
		view.c1.y = u.y;
		view.c2.y = u.z;
		view.c0.z = -f.x;
		view.c1.z = -f.y;
		view.c2.z = -f.z;
		view.c3.x = -dot( s, eye );
		view.c3.y = -dot( u, eye );
		view.c3.z = dot( f, eye );
		return view;
   }

   void PhotonMap::visualizePhotonMap(int width, int height)
   {
      std::cout << ">> Visualize Global Photon Map ...\n";
      uint8_t* image_buffer_ptr = nullptr;
      const size_t buffer_size = sizeof( uint8_t ) * width * height * 3;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&image_buffer_ptr), buffer_size ) );

      constexpr dim3 block(32, 32);
      const dim3 grid(divideUp( width, static_cast<int>(block.x) ), divideUp( height, static_cast<int>(block.y) ));
      cuVisualizePhotonMap<<<grid, block>>>(
         image_buffer_ptr,
         Device.GlobalPhotonsPtr, GlobalPhotonTree->getRoot(),
         Device.WorldBoundsPtr, Device.ToWorldsPtr,
         Device.VertexPtr, Device.NormalPtr, Device.IndexPtr, Device.VertexSizesPtr, Device.IndexSizesPtr,
         InverseViewMatrix, GlobalPhotonTree->getRootNode(), width, height, ObjectNum, MaxGlobalPhotonNum
      );
      CHECK_KERNEL;
      CHECK_CUDA( hipDeviceSynchronize() );

      auto* image_buffer = new uint8_t[width * height * 3];
      CHECK_CUDA( hipMemcpy( image_buffer, image_buffer_ptr, buffer_size, hipMemcpyDeviceToHost ) );
      hipFree( image_buffer_ptr );

      FIBITMAP* image = FreeImage_ConvertFromRawBits(
         image_buffer, width, height, width * 3, 24,
         FI_RGBA_RED_MASK, FI_RGBA_GREEN_MASK, FI_RGBA_BLUE_MASK, false
      );
      FreeImage_Save( FIF_PNG, image, "../global_photons.png" );
      FreeImage_Unload( image );
      delete [] image_buffer;
      std::cout << ">> Visualized Global Photon Map\n";
   }

   void PhotonMap::render(int width, int height)
   {
      std::cout << ">> Rendering ...\n";
      uint8_t* image_buffer_ptr = nullptr;
      const size_t buffer_size = sizeof( uint8_t ) * width * height * 3;
      CHECK_CUDA( hipMalloc( reinterpret_cast<void**>(&image_buffer_ptr), buffer_size ) );

      std::vector<uint> seed(1);
      std::seed_seq sequence{ std::chrono::system_clock::now().time_since_epoch().count() };
      sequence.generate( seed.begin(), seed.end() );

      constexpr dim3 block(16, 16);
      const dim3 grid(divideUp( width, static_cast<int>(block.x) ), divideUp( height, static_cast<int>(block.y) ));
      cuRenderScene<<<grid, block>>>(
         image_buffer_ptr,
         Device.GlobalPhotonsPtr, GlobalPhotonTree->getRoot(),
         Device.AreaLightsPtr, Device.MaterialsPtr, Device.WorldBoundsPtr, Device.ToWorldsPtr,
         Device.VertexPtr, Device.NormalPtr, Device.IndexPtr, Device.VertexSizesPtr, Device.IndexSizesPtr,
         InverseViewMatrix, GlobalPhotonTree->getRootNode(), width, height,
         LightNum, ObjectNum, MaxGlobalPhotonNum, seed[0]
      );
      CHECK_KERNEL;

      auto* image_buffer = new uint8_t[width * height * 3];
      CHECK_CUDA( hipMemcpy( image_buffer, image_buffer_ptr, buffer_size, hipMemcpyDeviceToHost ) );
      hipFree( image_buffer_ptr );

      FIBITMAP* image = FreeImage_ConvertFromRawBits(
         image_buffer, width, height, width * 3, 24,
         FI_RGBA_RED_MASK, FI_RGBA_GREEN_MASK, FI_RGBA_BLUE_MASK, false
      );
      FreeImage_Save( FIF_PNG, image, "../scene.png" );
      FreeImage_Unload( image );
      delete [] image_buffer;
      std::cout << ">> Rendered\n";
   }

   void PhotonMap::findNormals(
      std::vector<float3>& normals,
      const std::vector<float3>& vertices,
      const std::vector<int>& vertex_indices
   )
   {
      normals.resize( vertices.size() );
      const auto size = static_cast<int>(vertex_indices.size());
      for (int i = 0; i < size; i += 3) {
         const int n0 = vertex_indices[i];
         const int n1 = vertex_indices[i + 1];
         const int n2 = vertex_indices[i + 2];
         const float3 normal = cross( vertices[n1] - vertices[n0], vertices[n2] - vertices[n0] );
         normals[n0] += normal;
         normals[n1] += normal;
         normals[n2] += normal;
      }
      for (auto& n : normals) n = normalize( n );
   }

   void PhotonMap::readObjectFile(Box& box, const Mat& t, const std::string& file_path)
   {
      std::ifstream file(file_path);

      assert( file.is_open() );

      constexpr auto m = std::numeric_limits<float>::max();
      constexpr auto n = std::numeric_limits<float>::lowest();
      box.MinPoint = make_float3( m, m, m );
      box.MaxPoint = make_float3( n, n, n );

      bool found_normals = false;
      std::vector<float3> vertex_buffer, normal_buffer;
      std::vector<int> vertex_indices, normal_indices;
      while (!file.eof()) {
         std::string word;
         file >> word;

         if (word == "v") {
            float3 vertex;
            file >> vertex.x >> vertex.y >> vertex.z;
            vertex_buffer.emplace_back( vertex );

            const float3 v = transform( t, vertex );
            box.MinPoint.x = std::min( v.x, box.MinPoint.x );
            box.MinPoint.y = std::min( v.y, box.MinPoint.y );
            box.MinPoint.z = std::min( v.z, box.MinPoint.z );
            box.MaxPoint.x = std::max( v.x, box.MaxPoint.x );
            box.MaxPoint.y = std::max( v.y, box.MaxPoint.y );
            box.MaxPoint.z = std::max( v.z, box.MaxPoint.z );
         }
         else if (word == "vn") {
            float3 normal;
            file >> normal.x >> normal.y >> normal.z;
            normal_buffer.emplace_back( normal );
            found_normals = true;
         }
         else if (word == "f") {
            std::string face;
            const std::regex delimiter("[/]");
            for (int i = 0; i < 3; ++i) {
               file >> face;
               const std::sregex_token_iterator it(face.begin(), face.end(), delimiter, -1);
               const std::vector<std::string> vtn(it, std::sregex_token_iterator());
               vertex_indices.emplace_back( std::stoi( vtn[0] ) - 1 );
               if (found_normals && isNumber( vtn[2] )) {
                  normal_indices.emplace_back( std::stoi( vtn[2] ) - 1 );
                  found_normals = false;
               }
            }
         }
         else std::getline( file, word );
      }

      if (!found_normals) findNormals( normal_buffer, vertex_buffer, vertex_indices );

      VertexSizes.emplace_back( vertex_buffer.size() );
      IndexSizes.emplace_back( vertex_indices.size() );
      Vertices.insert(
         Vertices.end(),
         std::make_move_iterator( vertex_buffer.begin() ),
         std::make_move_iterator( vertex_buffer.end() )
      );
      Normals.insert(
         Normals.end(),
         std::make_move_iterator( normal_buffer.begin() ),
         std::make_move_iterator( normal_buffer.end() )
      );
      Indices.insert(
         Indices.end(),
         std::make_move_iterator( vertex_indices.begin() ),
         std::make_move_iterator( vertex_indices.end() )
      );
   }

   void PhotonMap::readObjectFile(
      std::vector<float3>& vertex_buffer,
      std::vector<float3>& normal_buffer,
      std::vector<int>& vertex_indices,
      const std::string& file_path
   )
   {
      std::ifstream file(file_path);

      assert( file.is_open() );

      bool found_normals = false;
      std::vector<int> normal_indices;
      while (!file.eof()) {
         std::string word;
         file >> word;

         if (word == "v") {
            float3 vertex;
            file >> vertex.x >> vertex.y >> vertex.z;
            vertex_buffer.emplace_back( vertex );
         }
         else if (word == "vn") {
            float3 normal;
            file >> normal.x >> normal.y >> normal.z;
            normal_buffer.emplace_back( normal );
            found_normals = true;
         }
         else if (word == "f") {
            std::string face;
            const std::regex delimiter("[/]");
            for (int i = 0; i < 3; ++i) {
               file >> face;
               const std::sregex_token_iterator it(face.begin(), face.end(), delimiter, -1);
               const std::vector<std::string> vtn(it, std::sregex_token_iterator());
               vertex_indices.emplace_back( std::stoi( vtn[0] ) - 1 );
               if (found_normals && isNumber( vtn[2] )) {
                  normal_indices.emplace_back( std::stoi( vtn[2] ) - 1 );
                  found_normals = false;
               }
            }
         }
         else std::getline( file, word );
      }

      if (!found_normals) findNormals( normal_buffer, vertex_buffer, vertex_indices );
   }

   // reference: https://paulbourke.net/dataformats/mtl/
   Material PhotonMap::getMaterial(const std::string& mtl_file_path)
   {
      std::ifstream file(mtl_file_path);

      assert( file.is_open() );

      Material material;
      while (!file.eof()) {
         std::string line;
         std::getline( file, line );

         const std::regex space_delimiter("[ ]");
         const std::sregex_token_iterator line_it(line.begin(), line.end(), space_delimiter, -1);
         const std::vector<std::string> parsed(line_it, std::sregex_token_iterator());
         if (parsed.empty()) continue;

         if (parsed[0] == "Ka") {
            material.Ambient.x = std::stof( parsed[1] );
            material.Ambient.y = std::stof( parsed[2] );
            material.Ambient.z = std::stof( parsed[3] );
         }
         else if (parsed[0] == "Kd") {
            material.Diffuse.x = std::stof( parsed[1] );
            material.Diffuse.y = std::stof( parsed[2] );
            material.Diffuse.z = std::stof( parsed[3] );
         }
         else if (parsed[0] == "Ks") {
            material.Specular.x = std::stof( parsed[1] );
            material.Specular.y = std::stof( parsed[2] );
            material.Specular.z = std::stof( parsed[3] );
         }
         else if (parsed[0] == "Ke") {
            material.Emission.x = std::stof( parsed[1] );
            material.Emission.y = std::stof( parsed[2] );
            material.Emission.z = std::stof( parsed[3] );
         }
         else if (parsed[0] == "Ns") material.SpecularExponent = std::stof( parsed[1] );
         else if (parsed[0] == "Ni") material.RefractiveIndex = std::stof( parsed[1] );
         else if (parsed[0] == "illum") {
            switch (std::stoi( parsed[1] )) {
               case 1:
                  material.UseAmbient = true;
                  material.UseDiffuse = true;
                  break;
               case 2:
                  material.UseAmbient = true;
                  material.UseDiffuse = true;
                  material.UseSpecular = true;
                  break;
               case 3: case 5:
                  material.UseAmbient = true;
                  material.UseSpecular = true;
                  material.UseReflectionRay = true;
                  break;
               case 4:
                  material.UseAmbient = true;
                  material.UseSpecular = true;
                  material.UseReflectionRay = true;
                  material.Transparent = true;
                  break;
               case 6: case 7:
                  material.UseAmbient = true;
                  material.UseSpecular = true;
                  material.UseReflectionRay = true;
                  material.UseRefractionRay = true;
                  material.Transparent = true;
                  break;
               default: break;
            }
         }
      }
      return material;
   }

   void PhotonMap::setObjects(const std::vector<std::tuple<std::string, std::string, Mat>>& objects)
   {
      Box box;
      for (size_t i = 0; i < objects.size(); ++i) {
         Materials.emplace_back( getMaterial( std::get<1>( objects[i] ) ) );
         ToWorlds.emplace_back( std::get<2>( objects[i] ) );
         readObjectFile( box, ToWorlds[i], std::get<0>( objects[i] ) );
         WorldBounds.emplace_back( box.MinPoint, box.MaxPoint );
      }
   }

   void PhotonMap::setLights(const std::vector<std::tuple<std::string, std::string, Mat>>& lights)
   {
      for (const auto& light : lights) {
         std::vector<int> vertex_indices;
         std::vector<float3> vertex_buffer, normal_buffer;
         readObjectFile( vertex_buffer, normal_buffer, vertex_indices, std::get<0>( light ) );
         const Material material = getMaterial( std::get<1>( light ) );
         const Mat m = std::get<2>( light );

         for (int i = 0; i < static_cast<int>(vertex_indices.size()); i += 3) {
            const int n0 = vertex_indices[i];
            const int n1 = vertex_indices[i + 1];
            const int n2 = vertex_indices[i + 2];
            const float3 v0 = transform( m, vertex_buffer[n0] );
            const float3 v1 = transform( m, vertex_buffer[n1] );
            const float3 v2 = transform( m, vertex_buffer[n2] );
            const float area = length( cross( v1 - v0, v2 - v0 ) ) * 0.5f;
            const float power = area *
               std::max( std::max( material.Emission.x, material.Emission.y ), material.Emission.z );
            TotalLightPower += power;
            AreaLights.emplace_back(
               area, power, material.Diffuse, material.Ambient, material.Emission,
               normal_buffer[n0], vertex_buffer[n0], vertex_buffer[n1], vertex_buffer[n2], m
            );
         }
      }
      LightNum = static_cast<int>(AreaLights.size());
   }
}
#endif